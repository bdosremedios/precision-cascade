#include "hip/hip_runtime.h"
#include "types/MatrixSparse/ImmutableMatrixSparse.h"

ImmutableMatrixSparse<float> ImmutableMatrixSparse<float>::operator*(const Scalar<float> &scalar) const {

    ImmutableMatrixSparse<float> created_mat(*this);

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            nnz,
            scalar.d_scalar, HIP_R_32F,
            created_mat.d_vals, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return created_mat;

}

ImmutableMatrixSparse<float> & ImmutableMatrixSparse<float>::operator*=(const Scalar<float> &scalar) {

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            nnz,
            scalar.d_scalar, HIP_R_32F,
            d_vals, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

ImmutableMatrixSparse<__half> ImmutableMatrixSparse<float>::to_half() const {

    ImmutableMatrixSparse<__half> created_mat(cu_handles, m_rows, n_cols, nnz);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(nnz)/static_cast<double>(NUM_THREADS))
    );

    check_cuda_error(hipMemcpy(
        created_mat.d_col_offsets,
        d_col_offsets,
        mem_size_col_offsets(),
        hipMemcpyDeviceToDevice
    ));

    check_cuda_error(hipMemcpy(
        created_mat.d_row_indices,
        d_row_indices,
        mem_size_row_indices(),
        hipMemcpyDeviceToDevice
    ));

    generalmatrix_sgl_kernels::cast_to_half<<<NUM_THREADS, NUM_BLOCKS>>>(
        d_vals, created_mat.d_vals, nnz
    );

    return created_mat;

}

ImmutableMatrixSparse<float> ImmutableMatrixSparse<float>::to_float() const {
    return ImmutableMatrixSparse<float>(*this);
}

ImmutableMatrixSparse<double> ImmutableMatrixSparse<float>::to_double() const {

    ImmutableMatrixSparse<double> created_mat(cu_handles, m_rows, n_cols, nnz);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(nnz)/static_cast<double>(NUM_THREADS))
    );

    check_cuda_error(hipMemcpy(
        created_mat.d_col_offsets,
        d_col_offsets,
        mem_size_col_offsets(),
        hipMemcpyDeviceToDevice
    ));

    check_cuda_error(hipMemcpy(
        created_mat.d_row_indices,
        d_row_indices,
        mem_size_row_indices(),
        hipMemcpyDeviceToDevice
    ));

    generalmatrix_sgl_kernels::cast_to_double<<<NUM_THREADS, NUM_BLOCKS>>>(
        d_vals, created_mat.d_vals, nnz
    );

    return created_mat;

}