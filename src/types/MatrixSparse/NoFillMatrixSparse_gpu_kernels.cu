#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include "types/MatrixSparse/NoFillMatrixSparse_gpu_kernels.cuh"

template <typename T>
__global__ void nofillmatrixsparse_kernels::update_pivot(
    int pivot_offset, int32_t *d_row_indices, T *d_vals, T *x_soln
) {
    int32_t row = d_row_indices[pivot_offset];
    x_soln[row] /= d_vals[pivot_offset];
}

template __global__ void nofillmatrixsparse_kernels::update_pivot<__half>(int, int32_t *, __half *, __half *);
template __global__ void nofillmatrixsparse_kernels::update_pivot<float>(int, int32_t *, float *, float *);
template __global__ void nofillmatrixsparse_kernels::update_pivot<double>(int, int32_t *, double *, double *);

template <typename T>
__global__ void nofillmatrixsparse_kernels::lowtri_update_remaining_col(
    int pivot_offset, int max_offset, int32_t *d_row_indices, T *d_vals, T *x_soln
) {
    __shared__ T xs;

    xs = x_soln[d_row_indices[pivot_offset]];

    if (pivot_offset+threadIdx.x+1 < max_offset) {
        int32_t row = d_row_indices[pivot_offset+threadIdx.x+1];
        x_soln[row] -= d_vals[pivot_offset+threadIdx.x+1]*xs;
    }

}

template __global__ void nofillmatrixsparse_kernels::lowtri_update_remaining_col<__half>(int, int, int32_t *, __half *, __half *);
template __global__ void nofillmatrixsparse_kernels::lowtri_update_remaining_col<float>(int, int, int32_t *, float *, float *);
template __global__ void nofillmatrixsparse_kernels::lowtri_update_remaining_col<double>(int, int, int32_t *, double *, double *);