#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <hip/hip_fp16.h>

#include "types/GeneralMatrix/GeneralMatrix_gpu_constants.cuh"

#include "types/MatrixSparse/NoFillMatrixSparse.h"

NoFillMatrixSparse<float>::NoFillMatrixSparse(const MatrixDense<float> &source_mat):
    NoFillMatrixSparse(source_mat, HIP_R_32F)
{}

NoFillMatrixSparse<float> NoFillMatrixSparse<float>::operator*(const Scalar<float> &scalar) const {

    NoFillMatrixSparse<float> created_mat(*this);

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            nnz,
            scalar.d_scalar, HIP_R_32F,
            created_mat.d_vals, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return created_mat;

}

NoFillMatrixSparse<float> & NoFillMatrixSparse<float>::operator*=(const Scalar<float> &scalar) {

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            nnz,
            scalar.d_scalar, HIP_R_32F,
            d_vals, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

Vector<float> NoFillMatrixSparse<float>::operator*(const Vector<float> &vec) const {

    if (vec.rows() != n_cols) {
        throw std::runtime_error(
            "NoFillMatrixSparse: invalid vec in operator*(const Vector<float> &vec)"
        );
    }

    Vector<float> new_vec(cu_handles, m_rows);

    hipsparseConstSpMatDescr_t spMatDescr;
    hipsparseConstDnVecDescr_t dnVecDescr_orig;
    hipsparseDnVecDescr_t dnVecDescr_new;
    
    check_cusparse_status(hipsparseCreateConstCsc(
        &spMatDescr,
        m_rows, n_cols, nnz,
        d_col_offsets, d_row_indices, d_vals,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
        HIP_R_32F
    ));
    check_cusparse_status(hipsparseCreateConstDnVec(&dnVecDescr_orig, n_cols, vec.d_vec, HIP_R_32F));
    check_cusparse_status(hipsparseCreateDnVec(&dnVecDescr_new, m_rows, new_vec.d_vec, HIP_R_32F));

    size_t bufferSize;
    check_cusparse_status(hipsparseSpMV_bufferSize(
        cu_handles.get_cusparse_handle(),
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        SCALAR_ONE_F.d_scalar, spMatDescr, dnVecDescr_orig,
        SCALAR_ZERO_F.d_scalar, dnVecDescr_new,
        HIP_R_32F,
        HIPSPARSE_SPMV_CSR_ALG1,
        &bufferSize
    ));

    float *d_buffer;
    check_cuda_error(hipMalloc(&d_buffer, bufferSize));

    check_cusparse_status(hipsparseSpMV(
        cu_handles.get_cusparse_handle(),
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        SCALAR_ONE_F.d_scalar, spMatDescr, dnVecDescr_orig,
        SCALAR_ZERO_F.d_scalar, dnVecDescr_new,
        HIP_R_32F,
        HIPSPARSE_SPMV_CSR_ALG1,
        d_buffer
    ));

    check_cuda_error(hipFree(d_buffer));
    
    check_cusparse_status(hipsparseDestroySpMat(spMatDescr));
    check_cusparse_status(hipsparseDestroyDnVec(dnVecDescr_orig));
    check_cusparse_status(hipsparseDestroyDnVec(dnVecDescr_new));

    return new_vec;

}

Vector<float> NoFillMatrixSparse<float>::transpose_prod(const Vector<float> &vec) const {

    if (vec.rows() != m_rows) {
        throw std::runtime_error(
            "NoFillMatrixSparse: invalid vec in transpose_prod"
        );
    }

    Vector<float> new_vec(cu_handles, n_cols);

    hipsparseConstSpMatDescr_t spMatDescr;
    hipsparseConstDnVecDescr_t dnVecDescr_orig;
    hipsparseDnVecDescr_t dnVecDescr_new;
    
    check_cusparse_status(hipsparseCreateConstCsc(
        &spMatDescr,
        m_rows, n_cols, nnz,
        d_col_offsets, d_row_indices, d_vals,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
        HIP_R_32F
    ));
    check_cusparse_status(hipsparseCreateConstDnVec(&dnVecDescr_orig, m_rows, vec.d_vec, HIP_R_32F));
    check_cusparse_status(hipsparseCreateDnVec(&dnVecDescr_new, n_cols, new_vec.d_vec, HIP_R_32F));

    size_t bufferSize;
    check_cusparse_status(hipsparseSpMV_bufferSize(
        cu_handles.get_cusparse_handle(),
        HIPSPARSE_OPERATION_TRANSPOSE,
        SCALAR_ONE_F.d_scalar, spMatDescr, dnVecDescr_orig,
        SCALAR_ZERO_F.d_scalar, dnVecDescr_new,
        HIP_R_32F,
        HIPSPARSE_SPMV_CSR_ALG1,
        &bufferSize
    ));

    float *d_buffer;
    check_cuda_error(hipMalloc(&d_buffer, bufferSize));

    check_cusparse_status(hipsparseSpMV(
        cu_handles.get_cusparse_handle(),
        HIPSPARSE_OPERATION_TRANSPOSE,
        SCALAR_ONE_F.d_scalar, spMatDescr, dnVecDescr_orig,
        SCALAR_ZERO_F.d_scalar, dnVecDescr_new,
        HIP_R_32F,
        HIPSPARSE_SPMV_CSR_ALG1,
        d_buffer
    ));

    check_cuda_error(hipFree(d_buffer));
    
    check_cusparse_status(hipsparseDestroySpMat(spMatDescr));
    check_cusparse_status(hipsparseDestroyDnVec(dnVecDescr_orig));
    check_cusparse_status(hipsparseDestroyDnVec(dnVecDescr_new));

    return new_vec;

}

NoFillMatrixSparse<__half> NoFillMatrixSparse<float>::to_half() const {

    NoFillMatrixSparse<__half> created_mat(cu_handles, m_rows, n_cols, nnz);

    double NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(nnz)/static_cast<double>(NUM_THREADS))
    );

    check_cuda_error(hipMemcpy(
        created_mat.d_col_offsets,
        d_col_offsets,
        mem_size_col_offsets(),
        hipMemcpyDeviceToDevice
    ));

    check_cuda_error(hipMemcpy(
        created_mat.d_row_indices,
        d_row_indices,
        mem_size_row_indices(),
        hipMemcpyDeviceToDevice
    ));

    generalmatrix_sgl_kernels::cast_to_half<<<NUM_THREADS, NUM_BLOCKS>>>(
        d_vals, created_mat.d_vals, nnz
    );
    check_kernel_launch(
        hipGetLastError(),
        "NoFillMatrixSparse<float>::to_half",
        "generalmatrix_sgl_kernels::cast_to_half",
        NUM_THREADS, NUM_BLOCKS
    );

    return created_mat;

}

NoFillMatrixSparse<float> NoFillMatrixSparse<float>::to_float() const {
    return NoFillMatrixSparse<float>(*this);
}

NoFillMatrixSparse<double> NoFillMatrixSparse<float>::to_double() const {

    NoFillMatrixSparse<double> created_mat(cu_handles, m_rows, n_cols, nnz);

    double NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(nnz)/static_cast<double>(NUM_THREADS))
    );

    check_cuda_error(hipMemcpy(
        created_mat.d_col_offsets,
        d_col_offsets,
        mem_size_col_offsets(),
        hipMemcpyDeviceToDevice
    ));

    check_cuda_error(hipMemcpy(
        created_mat.d_row_indices,
        d_row_indices,
        mem_size_row_indices(),
        hipMemcpyDeviceToDevice
    ));

    generalmatrix_sgl_kernels::cast_to_double<<<NUM_THREADS, NUM_BLOCKS>>>(
        d_vals, created_mat.d_vals, nnz
    );
    check_kernel_launch(
        hipGetLastError(),
        "NoFillMatrixSparse<float>::to_double",
        "generalmatrix_sgl_kernels::cast_to_double",
        NUM_THREADS, NUM_BLOCKS
    );

    return created_mat;

}