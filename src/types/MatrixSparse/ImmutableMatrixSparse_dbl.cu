#include "hip/hip_runtime.h"
#include "types/MatrixSparse/ImmutableMatrixSparse.h"

ImmutableMatrixSparse<double> ImmutableMatrixSparse<double>::operator*(const Scalar<double> &scalar) const {

    ImmutableMatrixSparse<double> created_mat(*this);

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            nnz,
            scalar.d_scalar, HIP_R_64F,
            created_mat.d_vals, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return created_mat;

}

ImmutableMatrixSparse<double> & ImmutableMatrixSparse<double>::operator*=(const Scalar<double> &scalar) {

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            nnz,
            scalar.d_scalar, HIP_R_64F,
            d_vals, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return *this;

}

ImmutableMatrixSparse<__half> ImmutableMatrixSparse<double>::to_half() const {

    ImmutableMatrixSparse<__half> created_mat(cu_handles, m_rows, n_cols, nnz);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(nnz)/static_cast<double>(NUM_THREADS))
    );

    check_cuda_error(hipMemcpy(
        created_mat.d_col_offsets,
        d_col_offsets,
        mem_size_col_offsets(),
        hipMemcpyDeviceToDevice
    ));

    check_cuda_error(hipMemcpy(
        created_mat.d_row_indices,
        d_row_indices,
        mem_size_row_indices(),
        hipMemcpyDeviceToDevice
    ));

    generalmatrix_dbl_kernels::cast_to_half<<<NUM_THREADS, NUM_BLOCKS>>>(
        d_vals, created_mat.d_vals, nnz
    );

    return created_mat;

}

ImmutableMatrixSparse<float> ImmutableMatrixSparse<double>::to_float() const {

    ImmutableMatrixSparse<float> created_mat(cu_handles, m_rows, n_cols, nnz);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(nnz)/static_cast<double>(NUM_THREADS))
    );

    check_cuda_error(hipMemcpy(
        created_mat.d_col_offsets,
        d_col_offsets,
        mem_size_col_offsets(),
        hipMemcpyDeviceToDevice
    ));

    check_cuda_error(hipMemcpy(
        created_mat.d_row_indices,
        d_row_indices,
        mem_size_row_indices(),
        hipMemcpyDeviceToDevice
    ));

    generalmatrix_dbl_kernels::cast_to_float<<<NUM_THREADS, NUM_BLOCKS>>>(
        d_vals, created_mat.d_vals, nnz
    );

    return created_mat;

}

ImmutableMatrixSparse<double> ImmutableMatrixSparse<double>::to_double() const {
    return ImmutableMatrixSparse<double>(*this);
}