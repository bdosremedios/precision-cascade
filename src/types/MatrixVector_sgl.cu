#include "hip/hip_runtime.h"
#include "types/MatrixVector.h"
#include "tools/cublas_check.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixVector<float> MatrixVector<float>::operator*(const float &scalar) const {

    MatrixVector<float> c(*this);

    hipblasStatus_t status = hipblasScalEx(
        handle, m, &scalar, HIP_R_32F, c.d_vec, HIP_R_32F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<float>& MatrixVector<float>::operator*=(const float &scalar) {

    hipblasStatus_t status = hipblasScalEx(
        handle, m, &scalar, HIP_R_32F, d_vec, HIP_R_32F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return *this;

}

template<>
MatrixVector<float> MatrixVector<float>::operator+(const MatrixVector<float> &vec) const {

    MatrixVector<float> c(*this);
    float alpha = 1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, c.d_vec, HIP_R_32F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<float> MatrixVector<float>::operator-(const MatrixVector<float> &vec) const {

    MatrixVector<float> c(*this);
    float alpha = -1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, c.d_vec, HIP_R_32F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<float>& MatrixVector<float>::operator+=(const MatrixVector<float> &vec) {

    float alpha = 1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, d_vec, HIP_R_32F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return *this;

}

template<>
MatrixVector<float>& MatrixVector<float>::operator-=(const MatrixVector<float> &vec) {

    float alpha = -1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, d_vec, HIP_R_32F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return *this;

}

template<>
float MatrixVector<float>::dot(const MatrixVector<float> &vec) const {
    
    float result;

    hipblasStatus_t status = hipblasDotEx(
        handle, m, d_vec, HIP_R_32F, 1, vec.d_vec, HIP_R_32F, 1, &result, HIP_R_32F, HIP_R_32F
    );
    check_cublas_status(status);

    return result;

}

template<>
float MatrixVector<float>::norm() const {

    float result;

    hipblasStatus_t status = hipblasNrm2Ex(
        handle, m, d_vec, HIP_R_32F, 1, &result, HIP_R_32F, HIP_R_32F
    );
    check_cublas_status(status);

    return result;

}