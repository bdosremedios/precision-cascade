#include "types/MatrixVector.h"
#include "tools/cublas_check.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixVector<float> MatrixVector<float>::operator*(const float &scalar) const {

    MatrixVector<float> c(*this);

    hipblasStatus_t status = hipblasScalEx(
        handle, m, &scalar, HIP_R_32F, c.d_vec, HIP_R_32F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<float>& MatrixVector<float>::operator*=(const float &scalar) {

    hipblasStatus_t status = hipblasScalEx(
        handle, m, &scalar, HIP_R_32F, d_vec, HIP_R_32F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return *this;

}