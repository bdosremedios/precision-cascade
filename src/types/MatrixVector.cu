#include "hip/hip_runtime.h"
#include "types/MatrixVector.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdexcept>
#include <string>

void check_cublas_status(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("cublas error: " + std::to_string(status));
    }
}

template<>
MatrixVector<double> MatrixVector<double>::operator*(const double &scalar) const {

    MatrixVector<double> c(*this);

    hipblasStatus_t status = hipblasScalEx(
        this->handle, m, &scalar, HIP_R_64F, c.d_vec, HIP_R_64F, 1, HIP_R_64F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<float> MatrixVector<float>::operator*(const float &scalar) const {

    MatrixVector<float> c(*this);

    hipblasStatus_t status = hipblasScalEx(
        handle, m, &scalar, HIP_R_32F, c.d_vec, HIP_R_32F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<__half> MatrixVector<__half>::operator*(const __half &scalar) const {

    MatrixVector<__half> c(*this);
    float *scalar_cast = static_cast<float *>(malloc(sizeof(float)));
    *scalar_cast = static_cast<float>(scalar);

    hipblasStatus_t status = hipblasScalEx(
        handle, m, scalar_cast, HIP_R_32F, c.d_vec, HIP_R_16F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    free(scalar_cast);

    return c;

}
