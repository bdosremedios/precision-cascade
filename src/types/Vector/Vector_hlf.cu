#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "types/Vector.h"

Vector<__half> Vector<__half>::operator*(const Scalar<__half> &scalar) const {

    Vector<__half> c(*this);

    Scalar<float> temp_cast(scalar.cast<float>());

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows,
            temp_cast.d_scalar, HIP_R_32F,
            c.d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return c;

}

Vector<__half> & Vector<__half>::operator*=(const Scalar<__half> &scalar) {

    Scalar<float> temp_cast(scalar.cast<float>());

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows,
            temp_cast.d_scalar, HIP_R_32F,
            d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

Vector<__half> Vector<__half>::operator+(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<__half> c(*this);
    Scalar<float> alpha(1.);

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows,
            alpha.d_scalar, HIP_R_32F,
            vec.d_vec, HIP_R_16F, 1,
            c.d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return c;

}

Vector<__half> Vector<__half>::operator-(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<__half> c(*this);
    Scalar<float> alpha(-1.);

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows,
            alpha.d_scalar, HIP_R_32F,
            vec.d_vec, HIP_R_16F, 1,
            c.d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return c;

}

Vector<__half> & Vector<__half>::operator+=(const Vector<__half> &vec) {

    check_vecvec_op_compatibility(vec);

    Scalar<float> alpha(1.);

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows,
            alpha.d_scalar, HIP_R_32F,
            vec.d_vec, HIP_R_16F, 1,
            d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

Vector<__half> & Vector<__half>::operator-=(const Vector<__half> &vec) {

    check_vecvec_op_compatibility(vec);

    Scalar<float> alpha(-1.);

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows,
            alpha.d_scalar, HIP_R_32F,
            vec.d_vec, HIP_R_16F, 1,
            d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

Scalar<__half> Vector<__half>::dot(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);
    
    Scalar<__half> result;

    check_cublas_status(
        hipblasDotEx(
            handle, m_rows,
            d_vec, HIP_R_16F, 1,
            vec.d_vec, HIP_R_16F, 1,
            result.d_scalar, HIP_R_16F,
            HIP_R_32F
        )
    );

    return result;

}

Scalar<__half> Vector<__half>::norm() const {

    Scalar<__half> result;

    check_cublas_status(
        hipblasNrm2Ex(
            handle, m_rows, d_vec, HIP_R_16F, 1, result.d_scalar, HIP_R_16F, HIP_R_32F
        )
    );

    return result;

}

namespace vec_hlf_kern
{
    __global__ void cast_to_float(__half *scalar_src, float *scalar_dest, int m) {
        int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (tid < m) {
            scalar_dest[tid] = __half2float(scalar_src[tid]);
        }
    }

    __global__ void cast_to_double(__half *scalar_src, double *scalar_dest, int m) {
        int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (tid < m) {
            scalar_dest[tid] = static_cast<double>(scalar_src[tid]);
        }
    }
}

Vector<__half> Vector<__half>::to_half() const { return Vector<__half>(*this); }

Vector<float> Vector<__half>::to_float() const {
    
    Vector<float> created_vec(handle, m_rows);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(m_rows)/static_cast<double>(NUM_THREADS))
    );
    vec_hlf_kern::cast_to_float<<<NUM_THREADS, NUM_BLOCKS>>>(d_vec, created_vec.d_vec, m_rows);

    return created_vec;

}

Vector<double> Vector<__half>::to_double() const {
    
    Vector<double> created_vec(handle, m_rows);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(m_rows)/static_cast<double>(NUM_THREADS))
    );
    vec_hlf_kern::cast_to_double<<<NUM_THREADS, NUM_BLOCKS>>>(d_vec, created_vec.d_vec, m_rows);

    return created_vec;

}