#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "types/Vector.h"

Vector<__half> Vector<__half>::operator*(const __half &scalar) const {

    Vector<__half> c(*this);
    float *scalar_cast = static_cast<float *>(malloc(sizeof(float)));
    *scalar_cast = static_cast<float>(scalar);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, scalar_cast, HIP_R_32F, c.d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );

    free(scalar_cast);

    return c;

}

Vector<__half> & Vector<__half>::operator*=(const __half &scalar) {

    float scalar_cast = static_cast<float>(scalar);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, &scalar_cast, HIP_R_32F, d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );


    return *this;

}

Vector<__half> Vector<__half>::operator+(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<__half> c(*this);
    float alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, c.d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );

    return c;

}

Vector<__half> Vector<__half>::operator-(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<__half> c(*this);
    float alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, c.d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );

    return c;

}

Vector<__half> & Vector<__half>::operator+=(const Vector<__half> &vec) {

    check_vecvec_op_compatibility(vec);

    float alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );

    return *this;

}

Vector<__half> & Vector<__half>::operator-=(const Vector<__half> &vec) {

    check_vecvec_op_compatibility(vec);

    float alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );

    return *this;

}

__half Vector<__half>::dot(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);
    
    __half result;

    check_cublas_status(
        hipblasDotEx(
            handle, m_rows, d_vec, HIP_R_16F, 1, vec.d_vec, HIP_R_16F, 1, &result, HIP_R_16F, HIP_R_32F
        )
    );

    return result;

}

__half Vector<__half>::norm() const {

    __half result;

    check_cublas_status(
        hipblasNrm2Ex(
            handle, m_rows, d_vec, HIP_R_16F, 1, &result, HIP_R_16F, HIP_R_32F
        )
    );

    return result;

}

namespace vec_hlf_kern
{
    __global__ void cast_to_float(__half *scalar_src, float *scalar_dest, int m) {
        int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (tid < m) {
            scalar_dest[tid] = __half2float(scalar_src[tid]);
        }
    }

    __global__ void cast_to_double(__half *scalar_src, double *scalar_dest, int m) {
        int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (tid < m) {
            scalar_dest[tid] = static_cast<double>(scalar_src[tid]);
        }
    }
}

Vector<__half> Vector<__half>::to_half() const { return Vector<__half>(*this); }

Vector<float> Vector<__half>::to_float() const {
    
    Vector<float> created_vec(handle, m_rows);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(m_rows)/static_cast<double>(NUM_THREADS))
    );
    vec_hlf_kern::cast_to_float<<<NUM_THREADS, NUM_BLOCKS>>>(d_vec, created_vec.d_vec, m_rows);

    return created_vec;

}

Vector<double> Vector<__half>::to_double() const {
    
    Vector<double> created_vec(handle, m_rows);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(m_rows)/static_cast<double>(NUM_THREADS))
    );
    vec_hlf_kern::cast_to_double<<<NUM_THREADS, NUM_BLOCKS>>>(d_vec, created_vec.d_vec, m_rows);

    return created_vec;

}