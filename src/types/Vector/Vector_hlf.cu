#include "hip/hip_runtime.h"
#include "types/Vector.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
Vector<__half> Vector<__half>::operator*(const __half &scalar) const {

    Vector<__half> c(*this);
    float *scalar_cast = static_cast<float *>(malloc(sizeof(float)));
    *scalar_cast = static_cast<float>(scalar);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, scalar_cast, HIP_R_32F, c.d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );

    free(scalar_cast);

    return c;

}

template<>
Vector<__half> & Vector<__half>::operator*=(const __half &scalar) {

    float scalar_cast = static_cast<float>(scalar);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, &scalar_cast, HIP_R_32F, d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );


    return *this;

}

template<>
Vector<__half> Vector<__half>::operator+(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<__half> c(*this);
    float alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, c.d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );

    return c;

}

template<>
Vector<__half> Vector<__half>::operator-(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<__half> c(*this);
    float alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, c.d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );

    return c;

}

template<>
Vector<__half> & Vector<__half>::operator+=(const Vector<__half> &vec) {

    check_vecvec_op_compatibility(vec);

    float alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );

    return *this;

}

template<>
Vector<__half> & Vector<__half>::operator-=(const Vector<__half> &vec) {

    check_vecvec_op_compatibility(vec);

    float alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, d_vec, HIP_R_16F, 1, HIP_R_32F
        )
    );

    return *this;

}

template<>
__half Vector<__half>::dot(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);
    
    __half result;

    check_cublas_status(
        hipblasDotEx(
            handle, m_rows, d_vec, HIP_R_16F, 1, vec.d_vec, HIP_R_16F, 1, &result, HIP_R_16F, HIP_R_32F
        )
    );

    return result;

}

template<>
__half Vector<__half>::norm() const {

    __half result;

    check_cublas_status(
        hipblasNrm2Ex(
            handle, m_rows, d_vec, HIP_R_16F, 1, &result, HIP_R_16F, HIP_R_32F
        )
    );

    return result;

}