#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "types/GeneralMatrix/GeneralMatrix_gpu_constants.cuh"

#include "types/Vector/Vector.h"

Vector<__half> Vector<__half>::operator*(const Scalar<__half> &scalar) const {

    Vector<__half> c(*this);

    Scalar<float> temp_cast(scalar.cast<float>());

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            temp_cast.d_scalar, HIP_R_32F,
            c.d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return c;

}

Vector<__half> & Vector<__half>::operator*=(const Scalar<__half> &scalar) {

    Scalar<float> temp_cast(scalar.cast<float>());

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            temp_cast.d_scalar, HIP_R_32F,
            d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

Vector<__half> Vector<__half>::operator+(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<__half> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            SCALAR_ONE_F.d_scalar, HIP_R_32F,
            vec.d_vec, HIP_R_16F, 1,
            c.d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return c;

}

Vector<__half> Vector<__half>::operator-(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<__half> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            SCALAR_MINUS_ONE_F.d_scalar, HIP_R_32F,
            vec.d_vec, HIP_R_16F, 1,
            c.d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return c;

}

Vector<__half> & Vector<__half>::operator+=(const Vector<__half> &vec) {

    check_vecvec_op_compatibility(vec);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            SCALAR_ONE_F.d_scalar, HIP_R_32F,
            vec.d_vec, HIP_R_16F, 1,
            d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

Vector<__half> & Vector<__half>::operator-=(const Vector<__half> &vec) {

    check_vecvec_op_compatibility(vec);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            SCALAR_MINUS_ONE_F.d_scalar, HIP_R_32F,
            vec.d_vec, HIP_R_16F, 1,
            d_vec, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

Scalar<__half> Vector<__half>::dot(const Vector<__half> &vec) const {

    check_vecvec_op_compatibility(vec);
    
    Scalar<__half> result;

    check_cublas_status(
        hipblasDotEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            d_vec, HIP_R_16F, 1,
            vec.d_vec, HIP_R_16F, 1,
            result.d_scalar, HIP_R_16F,
            HIP_R_32F
        )
    );

    return result;

}

Scalar<__half> Vector<__half>::norm() const {

    Scalar<__half> result;

    check_cublas_status(
        hipblasNrm2Ex(
            cu_handles.get_cublas_handle(),
            m_rows,
            d_vec, HIP_R_16F, 1,
            result.d_scalar, HIP_R_16F,
            HIP_R_32F
        )
    );

    return result;

}

Vector<__half> Vector<__half>::to_half() const { return Vector<__half>(*this); }

Vector<float> Vector<__half>::to_float() const {
    
    Vector<float> created_vec(cu_handles, m_rows);

    double NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(m_rows)/static_cast<double>(NUM_THREADS))
    );
    vector_hlf_kernels::cast_to_float<<<NUM_BLOCKS, NUM_THREADS>>>(d_vec, created_vec.d_vec, m_rows);
    check_kernel_launch(
        hipGetLastError(),
        "Vector<__half>::to_float",
        "vector_hlf_kernels::cast_to_float",
        NUM_BLOCKS, NUM_THREADS
    );

    return created_vec;

}

Vector<double> Vector<__half>::to_double() const {
    
    Vector<double> created_vec(cu_handles, m_rows);

    double NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(m_rows)/static_cast<double>(NUM_THREADS))
    );
    vector_hlf_kernels::cast_to_double<<<NUM_BLOCKS, NUM_THREADS>>>(d_vec, created_vec.d_vec, m_rows);
    check_kernel_launch(
        hipGetLastError(),
        "Vector<__half>::to_double",
        "vector_hlf_kernels::cast_to_double",
        NUM_BLOCKS, NUM_THREADS
    );

    return created_vec;

}