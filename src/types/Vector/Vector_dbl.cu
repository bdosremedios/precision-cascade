#include "hip/hip_runtime.h"
#include <cmath>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "types/Vector/Vector.h"

Vector<double> Vector<double>::operator*(const Scalar<double> &scalar) const {

    Vector<double> c(*this);

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            scalar.d_scalar, HIP_R_64F,
            c.d_vec, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

Vector<double> & Vector<double>::operator*=(const Scalar<double> &scalar) {

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            scalar.d_scalar, HIP_R_64F,
            d_vec, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return *this;

}

Vector<double> Vector<double>::operator+(const Vector<double> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<double> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            SCALAR_ONE_D.d_scalar, HIP_R_64F,
            vec.d_vec, HIP_R_64F, 1,
            c.d_vec, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

Vector<double> Vector<double>::operator-(const Vector<double> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<double> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            SCALAR_MINUS_ONE_D.d_scalar, HIP_R_64F,
            vec.d_vec, HIP_R_64F, 1,
            c.d_vec, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

Vector<double> & Vector<double>::operator+=(const Vector<double> &vec) {

    check_vecvec_op_compatibility(vec);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            SCALAR_ONE_D.d_scalar, HIP_R_64F,
            vec.d_vec, HIP_R_64F, 1,
            d_vec, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return *this;

}

Vector<double> & Vector<double>::operator-=(const Vector<double> &vec) {

    check_vecvec_op_compatibility(vec);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            SCALAR_MINUS_ONE_D.d_scalar, HIP_R_64F,
            vec.d_vec, HIP_R_64F, 1,
            d_vec, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return *this;

}

Scalar<double> Vector<double>::dot(const Vector<double> &vec) const {

    check_vecvec_op_compatibility(vec);
    
    Scalar<double> result;

    check_cublas_status(
        hipblasDotEx(
            cu_handles.get_cublas_handle(),
            m_rows,
            d_vec, HIP_R_64F, 1,
            vec.d_vec, HIP_R_64F, 1,
            result.d_scalar, HIP_R_64F,
            HIP_R_64F
        )
    );

    return result;

}

Scalar<double> Vector<double>::norm() const {

    Scalar<double> result;

    check_cublas_status(
        hipblasNrm2Ex(
            cu_handles.get_cublas_handle(),
            m_rows,
            d_vec, HIP_R_64F, 1,
            result.d_scalar, HIP_R_64F,
            HIP_R_64F
        )
    );

    return result;

}

Vector<__half> Vector<double>::to_half() const {

    Vector<__half> created_vec(cu_handles, m_rows);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(m_rows)/static_cast<double>(NUM_THREADS))
    );
    vector_dbl_kernels::cast_to_half<<<NUM_THREADS, NUM_BLOCKS>>>(d_vec, created_vec.d_vec, m_rows);

    return created_vec;

}

Vector<float> Vector<double>::to_float() const {

    Vector<float> created_vec(cu_handles, m_rows);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(m_rows)/static_cast<double>(NUM_THREADS))
    );
    vector_dbl_kernels::cast_to_float<<<NUM_THREADS, NUM_BLOCKS>>>(d_vec, created_vec.d_vec, m_rows);

    return created_vec;

}

Vector<double> Vector<double>::to_double() const { return Vector<double>(*this); }