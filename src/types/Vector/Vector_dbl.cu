#include "hip/hip_runtime.h"
#include "types/Vector.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
Vector<double> Vector<double>::operator*(const double &scalar) const {

    Vector<double> c(*this);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, &scalar, HIP_R_64F, c.d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return c;

}

template<>
Vector<double> & Vector<double>::operator*=(const double &scalar) {

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, &scalar, HIP_R_64F, d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return *this;

}

template<>
Vector<double> Vector<double>::operator+(const Vector<double> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<double> c(*this);
    double alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, c.d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return c;

}

template<>
Vector<double> Vector<double>::operator-(const Vector<double> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<double> c(*this);
    double alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, c.d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return c;

}

template<>
Vector<double> & Vector<double>::operator+=(const Vector<double> &vec) {

    check_vecvec_op_compatibility(vec);

    double alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return *this;

}

template<>
Vector<double> & Vector<double>::operator-=(const Vector<double> &vec) {

    check_vecvec_op_compatibility(vec);

    double alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return *this;

}

template<>
double Vector<double>::dot(const Vector<double> &vec) const {

    check_vecvec_op_compatibility(vec);
    
    double result;

    check_cublas_status(
        hipblasDotEx(
            handle, m_rows, d_vec, HIP_R_64F, 1, vec.d_vec, HIP_R_64F, 1, &result, HIP_R_64F, HIP_R_64F
        )
    );

    return result;

}

template<>
double Vector<double>::norm() const {

    double result;

    check_cublas_status(
        hipblasNrm2Ex(
            handle, m_rows, d_vec, HIP_R_64F, 1, &result, HIP_R_64F, HIP_R_64F
        )
    );

    return result;

}