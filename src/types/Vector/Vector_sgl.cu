#include "hip/hip_runtime.h"
#include "types/Vector.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
Vector<float> Vector<float>::operator*(const float &scalar) const {

    Vector<float> c(*this);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, &scalar, HIP_R_32F, c.d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return c;

}

template<>
Vector<float> & Vector<float>::operator*=(const float &scalar) {

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, &scalar, HIP_R_32F, d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return *this;

}

template<>
Vector<float> Vector<float>::operator+(const Vector<float> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<float> c(*this);
    float alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, c.d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return c;

}

template<>
Vector<float> Vector<float>::operator-(const Vector<float> &vec) const {

    check_vecvec_op_compatibility(vec);

    Vector<float> c(*this);
    float alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, c.d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return c;

}

template<>
Vector<float> & Vector<float>::operator+=(const Vector<float> &vec) {

    check_vecvec_op_compatibility(vec);

    float alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return *this;

}

template<>
Vector<float> & Vector<float>::operator-=(const Vector<float> &vec) {

    check_vecvec_op_compatibility(vec);

    float alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return *this;

}

template<>
float Vector<float>::dot(const Vector<float> &vec) const {

    check_vecvec_op_compatibility(vec);
    
    float result;

    check_cublas_status(
        hipblasDotEx(
            handle, m_rows, d_vec, HIP_R_32F, 1, vec.d_vec, HIP_R_32F, 1, &result, HIP_R_32F, HIP_R_32F
        )
    );

    return result;

}

template<>
float Vector<float>::norm() const {

    float result;

    check_cublas_status(
        hipblasNrm2Ex(
            handle, m_rows, d_vec, HIP_R_32F, 1, &result, HIP_R_32F, HIP_R_32F
        )
    );

    return result;

}