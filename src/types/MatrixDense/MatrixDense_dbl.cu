#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include "types/GeneralMatrix/GeneralMatrix_gpu_constants.cuh"

#include "types/MatrixDense/MatrixDense.h"

MatrixDense<double> MatrixDense<double>::operator*(const Scalar<double> &scalar) const {

    MatrixDense<double> c(*this);

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            scalar.d_scalar, HIP_R_64F,
            c.d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

MatrixDense<double> & MatrixDense<double>::operator*=(const Scalar<double> &scalar) {

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            scalar.d_scalar, HIP_R_64F,
            d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return *this;

}

Vector<double> MatrixDense<double>::operator*(const Vector<double> &vec) const {

    if (vec.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid vec in matrix-vector prod (operator*(const Vector<double> &vec))"
        );
    }

    Vector<double> c(Vector<double>::Zero(cu_handles, m_rows));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, n_cols,
            SCALAR_ONE_D.d_scalar,
            d_mat, HIP_R_64F, m_rows,
            vec.d_vec, HIP_R_64F, n_cols,
            SCALAR_ZERO_D.d_scalar,
            c.d_vec, HIP_R_64F, m_rows,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

Vector<double> MatrixDense<double>::transpose_prod(const Vector<double> &vec) const {

    if (vec.rows() != m_rows) { throw std::runtime_error("MatrixDense: invalid vec in transpose_prod"); }

    Vector<double> c(Vector<double>::Zero(cu_handles, n_cols));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_T, HIPBLAS_OP_N,
            n_cols, 1, m_rows,
            SCALAR_ONE_D.d_scalar,
            d_mat, HIP_R_64F, m_rows,
            vec.d_vec, HIP_R_64F, m_rows,
            SCALAR_ZERO_D.d_scalar,
            c.d_vec, HIP_R_64F, n_cols,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

MatrixDense<double> MatrixDense<double>::operator*(const MatrixDense<double> &mat) const {

    if (mat.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix-matrix prod (operator*(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<double> c(MatrixDense<double>::Zero(cu_handles, m_rows, mat.cols()));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, mat.cols(), n_cols,
            SCALAR_ONE_D.d_scalar,
            d_mat, HIP_R_64F, m_rows,
            mat.d_mat, HIP_R_64F, n_cols,
            SCALAR_ZERO_D.d_scalar,
            c.d_mat, HIP_R_64F, m_rows,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

MatrixDense<double> MatrixDense<double>::operator+(const MatrixDense<double> &mat) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix add (operator+(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<double> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            SCALAR_ONE_D.d_scalar, HIP_R_64F,
            mat.d_mat, HIP_R_64F, 1,
            c.d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

MatrixDense<double> MatrixDense<double>::operator-(const MatrixDense<double> &mat) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix subtract (operator-(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<double> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            SCALAR_MINUS_ONE_D.d_scalar, HIP_R_64F,
            mat.d_mat, HIP_R_64F, 1,
            c.d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

Scalar<double> MatrixDense<double>::norm() const {

    Scalar<double> result;

    check_cublas_status(
        hipblasNrm2Ex(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            d_mat, HIP_R_64F, 1,
            result.d_scalar, HIP_R_64F,
            HIP_R_64F
        )
    );

    return result;

}

MatrixDense<__half> MatrixDense<double>::to_half() const {

    MatrixDense<__half> created_mat(cu_handles, m_rows, n_cols);

    int NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    int NUM_BLOCKS = std::ceil(static_cast<double>(m_rows*n_cols)/static_cast<double>(NUM_THREADS));

    if (NUM_BLOCKS > 0) {

        generalmatrix_dbl_kernels::cast_to_half<<<NUM_BLOCKS, NUM_THREADS>>>(
            d_mat, created_mat.d_mat, m_rows*n_cols
        );

        check_kernel_launch(
            hipGetLastError(),
            "MatrixDense<double>::to_half",
            "generalmatrix_dbl_kernels::cast_to_half",
            NUM_BLOCKS, NUM_THREADS
        );

    }

    return created_mat;

}

MatrixDense<float> MatrixDense<double>::to_float() const {

    MatrixDense<float> created_mat(cu_handles, m_rows, n_cols);

    int NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    int NUM_BLOCKS = std::ceil(static_cast<double>(m_rows*n_cols)/static_cast<double>(NUM_THREADS));

    if (NUM_BLOCKS > 0) {

        generalmatrix_dbl_kernels::cast_to_float<<<NUM_BLOCKS, NUM_THREADS>>>(
            d_mat, created_mat.d_mat, m_rows*n_cols
        );

        check_kernel_launch(
            hipGetLastError(),
            "MatrixDense<double>::to_float",
            "generalmatrix_dbl_kernels::cast_to_float",
            NUM_BLOCKS, NUM_THREADS
        );

    }

    return created_mat;

}

MatrixDense<double> MatrixDense<double>::to_double() const {
    return MatrixDense<double>(*this);
}