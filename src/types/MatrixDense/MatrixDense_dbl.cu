#include "types/MatrixDense.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixDense<double> MatrixDense<double>::operator*(const double &scalar) const {

    MatrixDense<double> c(*this);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows*n_cols, &scalar, HIP_R_64F, c.d_mat, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return c;
}