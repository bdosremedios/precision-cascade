#include "hip/hip_runtime.h"
#include "types/GeneralMatrix/GeneralMatrix_gpu_constants.cuh"
#include "types/MatrixDense/MatrixDense.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

template <>
MatrixDense<double> MatrixDense<double>::operator*(
    const Scalar<double> &scalar
) const {

    MatrixDense<double> c(*this);

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            scalar.d_scalar, HIP_R_64F,
            c.d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

template <>
MatrixDense<double> & MatrixDense<double>::operator*=(
    const Scalar<double> &scalar
) {

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            scalar.d_scalar, HIP_R_64F,
            d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return *this;

}

template <>
Vector<double> MatrixDense<double>::operator*(
    const Vector<double> &vec
) const {

    if (vec.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid vec in matrix-vector prod "
            "(operator*(const Vector<double> &vec))"
        );
    }

    Vector<double> c(Vector<double>::Zero(cu_handles, m_rows));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, n_cols,
            SCALAR_ONE_D.d_scalar,
            d_mat, HIP_R_64F, m_rows,
            vec.d_vec, HIP_R_64F, n_cols,
            SCALAR_ZERO_D.d_scalar,
            c.d_vec, HIP_R_64F, m_rows,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

template <>
Vector<double> MatrixDense<double>::mult_subset_cols(
    int start, int cols, const Vector<double> &vec
) const {

    if ((start < 0) || ((start+cols) > n_cols) || (cols <= 0)) {
        throw std::runtime_error(
            "MatrixDense: invalid column values in mult_subset_cols"
        );
    }
    if (vec.rows() != cols) {
        throw std::runtime_error(
            "MatrixDense: invalid vec in mult_subset_cols"
        );
    }

    Vector<double> c(Vector<double>::Zero(cu_handles, m_rows));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, cols,
            SCALAR_ONE_D.d_scalar,
            d_mat+start*m_rows, HIP_R_64F, m_rows,
            vec.d_vec, HIP_R_64F, n_cols,
            SCALAR_ZERO_D.d_scalar,
            c.d_vec, HIP_R_64F, m_rows,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

template <>
Vector<double> MatrixDense<double>::transpose_prod(
    const Vector<double> &vec
) const {

    if (vec.rows() != m_rows) {
        throw std::runtime_error("MatrixDense: invalid vec in transpose_prod");
    }

    Vector<double> c(Vector<double>::Zero(cu_handles, n_cols));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_T, HIPBLAS_OP_N,
            n_cols, 1, m_rows,
            SCALAR_ONE_D.d_scalar,
            d_mat, HIP_R_64F, m_rows,
            vec.d_vec, HIP_R_64F, m_rows,
            SCALAR_ZERO_D.d_scalar,
            c.d_vec, HIP_R_64F, n_cols,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

template <>
Vector<double> MatrixDense<double>::transpose_prod_subset_cols(
    int start, int cols, const Vector<double> &vec
) const {

    if ((start < 0) || ((start+cols) > n_cols) || (cols <= 0)) {
        throw std::runtime_error(
            "MatrixDense: invalid column values in transpose_prod_subset_cols"
        );
    }
    if (vec.rows() != m_rows) {
        throw std::runtime_error(
            "MatrixDense: invalid vec in transpose_prod_subset_cols"
        );
    }

    Vector<double> c(Vector<double>::Zero(cu_handles, cols));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_T, HIPBLAS_OP_N,
            cols, 1, m_rows,
            SCALAR_ONE_D.d_scalar,
            d_mat+start*m_rows, HIP_R_64F, m_rows,
            vec.d_vec, HIP_R_64F, m_rows,
            SCALAR_ZERO_D.d_scalar,
            c.d_vec, HIP_R_64F, cols,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

template <>
MatrixDense<double> MatrixDense<double>::operator*(
    const MatrixDense<double> &mat
) const {

    if (mat.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix-matrix prod "
            "(operator*(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<double> c(
        MatrixDense<double>::Zero(cu_handles, m_rows, mat.cols())
    );

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, mat.cols(), n_cols,
            SCALAR_ONE_D.d_scalar,
            d_mat, HIP_R_64F, m_rows,
            mat.d_mat, HIP_R_64F, n_cols,
            SCALAR_ZERO_D.d_scalar,
            c.d_mat, HIP_R_64F, m_rows,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

template <>
MatrixDense<double> MatrixDense<double>::operator+(
    const MatrixDense<double> &mat
) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix add "
            "(operator+(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<double> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            SCALAR_ONE_D.d_scalar, HIP_R_64F,
            mat.d_mat, HIP_R_64F, 1,
            c.d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

template <>
MatrixDense<double> MatrixDense<double>::operator-(
    const MatrixDense<double> &mat
) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix subtract "
            "(operator-(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<double> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            SCALAR_MINUS_ONE_D.d_scalar, HIP_R_64F,
            mat.d_mat, HIP_R_64F, 1,
            c.d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

template <>
Scalar<double> MatrixDense<double>::norm() const {

    Scalar<double> result;

    check_cublas_status(
        hipblasNrm2Ex(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            d_mat, HIP_R_64F, 1,
            result.d_scalar, HIP_R_64F,
            HIP_R_64F
        )
    );

    return result;

}

template <>
MatrixDense<__half> MatrixDense<double>::to_half() const {

    MatrixDense<__half> created_mat(cu_handles, m_rows, n_cols);

    int NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    int NUM_BLOCKS = std::ceil(
        static_cast<double>(m_rows*n_cols) /
        static_cast<double>(NUM_THREADS)
    );

    if (NUM_BLOCKS > 0) {

        generalmatrix_dbl_kernels::cast_to_half<<<NUM_BLOCKS, NUM_THREADS>>>(
            d_mat, created_mat.d_mat, m_rows*n_cols
        );

        check_kernel_launch(
            hipGetLastError(),
            "MatrixDense<double>::to_half",
            "generalmatrix_dbl_kernels::cast_to_half",
            NUM_BLOCKS, NUM_THREADS
        );

    }

    return created_mat;

}

template <>
MatrixDense<float> MatrixDense<double>::to_float() const {

    MatrixDense<float> created_mat(cu_handles, m_rows, n_cols);

    int NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    int NUM_BLOCKS = std::ceil(
        static_cast<double>(m_rows*n_cols) /
        static_cast<double>(NUM_THREADS)
    );

    if (NUM_BLOCKS > 0) {

        generalmatrix_dbl_kernels::cast_to_float<<<NUM_BLOCKS, NUM_THREADS>>>(
            d_mat, created_mat.d_mat, m_rows*n_cols
        );

        check_kernel_launch(
            hipGetLastError(),
            "MatrixDense<double>::to_float",
            "generalmatrix_dbl_kernels::cast_to_float",
            NUM_BLOCKS, NUM_THREADS
        );

    }

    return created_mat;

}

template <>
MatrixDense<double> MatrixDense<double>::to_double() const {
    return MatrixDense<double>(*this);
}