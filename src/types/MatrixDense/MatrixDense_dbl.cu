#include "hip/hip_runtime.h"
#include "types/MatrixDense.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

MatrixDense<double> MatrixDense<double>::operator*(const Scalar<double> &scalar) const {

    MatrixDense<double> c(*this);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows*n_cols,
            scalar.d_scalar, HIP_R_64F,
            c.d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;
}

Vector<double> MatrixDense<double>::operator*(const Vector<double> &vec) const {

    if (vec.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid vec in matrix-vector prod (operator*(const Vector<double> &vec))"
        );
    }

    Vector<double> c(Vector<double>::Zero(handle, m_rows));

    check_cublas_status(
        hipblasGemmEx(
            handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, n_cols,
            SCALAR_ONE_D.d_scalar,
            d_mat, HIP_R_64F, m_rows,
            vec.d_vec, HIP_R_64F, n_cols,
            SCALAR_ZERO_D.d_scalar,
            c.d_vec, HIP_R_64F, m_rows,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

Vector<double> MatrixDense<double>::transpose_prod(const Vector<double> &vec) const {

    if (vec.rows() != m_rows) { throw std::runtime_error("MatrixDense: invalid vec in transpose_prod"); }

    Vector<double> c(Vector<double>::Zero(handle, n_cols));

    check_cublas_status(
        hipblasGemmEx(
            handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            n_cols, 1, m_rows,
            SCALAR_ONE_D.d_scalar,
            d_mat, HIP_R_64F, m_rows,
            vec.d_vec, HIP_R_64F, m_rows,
            SCALAR_ZERO_D.d_scalar,
            c.d_vec, HIP_R_64F, n_cols,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

MatrixDense<double> MatrixDense<double>::operator*(const MatrixDense<double> &mat) const {

    if (mat.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix-matrix prod (operator*(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<double> c(MatrixDense<double>::Zero(handle, m_rows, mat.cols()));

    check_cublas_status(
        hipblasGemmEx(
            handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, mat.cols(), n_cols,
            SCALAR_ONE_D.d_scalar,
            d_mat, HIP_R_64F, m_rows,
            mat.d_mat, HIP_R_64F, n_cols,
            SCALAR_ZERO_D.d_scalar,
            c.d_mat, HIP_R_64F, m_rows,
            HIPBLAS_COMPUTE_64F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

MatrixDense<double> MatrixDense<double>::operator+(const MatrixDense<double> &mat) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix add (operator+(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<double> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows*n_cols,
            SCALAR_ONE_D.d_scalar, HIP_R_64F,
            mat.d_mat, HIP_R_64F, 1,
            c.d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

MatrixDense<double> MatrixDense<double>::operator-(const MatrixDense<double> &mat) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix subtract (operator-(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<double> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows*n_cols,
            SCALAR_MINUS_ONE_D.d_scalar, HIP_R_64F,
            mat.d_mat, HIP_R_64F, 1,
            c.d_mat, HIP_R_64F, 1,
            HIP_R_64F
        )
    );

    return c;

}

Scalar<double> MatrixDense<double>::norm() const {

    Scalar<double> result;

    check_cublas_status(
        hipblasNrm2Ex(
            handle, m_rows*n_cols,
            d_mat, HIP_R_64F, 1,
            result.d_scalar, HIP_R_64F,
            HIP_R_64F
        )
    );

    return result;

}

namespace matdense_dbl_kern
{
    __global__ void cast_to_half(double *mat_src, half *mat_dest, int m) {
        int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (tid < m) {
            mat_dest[tid] = __double2half(mat_src[tid]);
        }
    }

    __global__ void cast_to_float(double *mat_src, float *mat_dest, int m) {
        int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
        if (tid < m) {
            mat_dest[tid] = __double2float_rn(mat_src[tid]);
        }
    }
}

MatrixDense<__half> MatrixDense<double>::to_half() const {

    MatrixDense<__half> created_mat(handle, m_rows, n_cols);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(m_rows*n_cols)/static_cast<double>(NUM_THREADS))
    );
    matdense_dbl_kern::cast_to_half<<<NUM_THREADS, NUM_BLOCKS>>>(d_mat, created_mat.d_mat, m_rows*n_cols);

    return created_mat;

}

MatrixDense<float> MatrixDense<double>::to_float() const {

    MatrixDense<float> created_mat(handle, m_rows, n_cols);

    double NUM_THREADS = 1024; // threads per thread block just 1 warp
    double NUM_BLOCKS = static_cast<double>(
        std::ceil(static_cast<double>(m_rows*n_cols)/static_cast<double>(NUM_THREADS))
    );
    matdense_dbl_kern::cast_to_float<<<NUM_THREADS, NUM_BLOCKS>>>(d_mat, created_mat.d_mat, m_rows*n_cols);

    return created_mat;

}

MatrixDense<double> MatrixDense<double>::to_double() const { return MatrixDense<double>(*this); }