#include "hip/hip_runtime.h"
#include "types/MatrixDense/MatrixDense_gpu_kernels.cuh"

template <typename T>
__global__ void matrixdense_kernels::lowtri_blk_solve_warp(
    const T *L, int m_rows, int diag_offset, T *x_soln
) {

    volatile __shared__ T xs;

    #pragma unroll
    for (int i=0; i<WARPSIZE; ++i) {

        if (diag_offset+threadIdx.x < m_rows) {

            if (threadIdx.x == i) {
                xs = x_soln[diag_offset+threadIdx.x]/L[(diag_offset+threadIdx.x)+(diag_offset+threadIdx.x)*m_rows];
                x_soln[diag_offset+threadIdx.x] = xs;
            }

            if (threadIdx.x >= i+1) {
                x_soln[diag_offset+threadIdx.x] -= (
                    static_cast<T>(L[(diag_offset+threadIdx.x)+(diag_offset+i)*m_rows])*static_cast<T>(xs)
                );
            }

        }

    }

}

template __global__ void matrixdense_kernels::lowtri_blk_solve_warp<__half>(const __half *, int , int , __half *);
template __global__ void matrixdense_kernels::lowtri_blk_solve_warp<float>(const float *, int , int , float *);
template __global__ void matrixdense_kernels::lowtri_blk_solve_warp<double>(const double *, int , int , double *);

template <typename T>
__global__ void matrixdense_kernels::lowtri_rect_update_warp(const T *L, int m_rows, int diag_offset, T *x_soln) {

    int soln_row = diag_offset + threadIdx.x; 
    int col = diag_offset + threadIdx.x;
    int row = diag_offset + WARPSIZE + (blockIdx.y*blockDim.y) + threadIdx.y;

    if (row < m_rows) {
        atomicAdd(x_soln+row, -L[row+col*m_rows]*x_soln[soln_row]);
    }

}

template __global__ void matrixdense_kernels::lowtri_rect_update_warp<__half>(const __half *, int , int , __half *);
template __global__ void matrixdense_kernels::lowtri_rect_update_warp<float>(const float *, int , int , float *);
template __global__ void matrixdense_kernels::lowtri_rect_update_warp<double>(const double *, int , int , double *);

// *** MatrixDense double kernel implementations ***

__global__ void matrixdense_dbl_kernels::solve_pivot_and_find_alpha(double *rhs, double *diag, double *alpha) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    rhs[tid] = rhs[tid]/diag[tid];
    alpha[tid] = -rhs[tid];
}

__global__ void matrixdense_dbl_kernels::fwrd_blk_solve(const double *L, int m_rows, int diag_offset, double *x_soln) {

    volatile __shared__ double xs;

    #pragma unroll
    for (int i=0; i<32; ++i) {

        if (diag_offset+threadIdx.x < m_rows) {

            if (threadIdx.x == i) {
                xs = x_soln[diag_offset+threadIdx.x]/L[(diag_offset+threadIdx.x)+(diag_offset+threadIdx.x)*m_rows];
                x_soln[diag_offset+threadIdx.x] = xs;
            }

            if (threadIdx.x >= i+1) {
                x_soln[diag_offset+threadIdx.x] -= L[(diag_offset+threadIdx.x)+(diag_offset+i)*m_rows]*xs;
            }

        }

    }

}

__global__ void matrixdense_dbl_kernels::fwrd_rect_update(const double *L, int m_rows, int diag_offset, double *x_soln) {

    int soln_row = diag_offset + threadIdx.x; 
    int col = diag_offset + threadIdx.x;
    int row = diag_offset + 32 + (blockIdx.y*blockDim.y) + threadIdx.y;

    if (row < m_rows) {
        atomicAdd(x_soln+row, -L[row+col*m_rows]*x_soln[soln_row]);
    }

}

// *** MatrixDense single kernel implementations ***

__global__ void matrixdense_sgl_kernels::solve_pivot_and_find_alpha(float *rhs, float *diag, float *alpha) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    rhs[tid] = rhs[tid]/diag[tid];
    alpha[tid] = -rhs[tid];
}

// *** MatrixDense half kernel implementations ***

__global__ void matrixdense_hlf_kernels::solve_pivot_and_find_alpha(__half *rhs, __half *diag, float *alpha) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    rhs[tid] = rhs[tid]/diag[tid];
    alpha[tid] = __half2float(-rhs[tid]);
}