#include "hip/hip_runtime.h"
#include "types/MatrixDense.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixDense<__half> MatrixDense<__half>::operator*(const __half &scalar) const {

    MatrixDense<__half> c(*this);

    float scalar_cast = static_cast<float>(scalar);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows*n_cols, &scalar_cast, HIP_R_32F, c.d_mat, HIP_R_16F, 1, HIP_R_32F
        )
    );

    return c;
}