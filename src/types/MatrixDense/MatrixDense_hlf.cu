#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include "types/GeneralMatrix/GeneralMatrix_gpu_constants.cuh"

#include "types/MatrixDense/MatrixDense.h"

MatrixDense<__half> MatrixDense<__half>::operator*(const Scalar<__half> &scalar) const {

    MatrixDense<__half> c(*this);

    Scalar<float> temp_cast(scalar.cast<float>());

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            temp_cast.d_scalar, HIP_R_32F,
            c.d_mat, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return c;

}

MatrixDense<__half> & MatrixDense<__half>::operator*=(const Scalar<__half> &scalar) {

    Scalar<float> temp_cast(scalar.cast<float>());

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            temp_cast.d_scalar, HIP_R_32F,
            d_mat, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

Vector<__half> MatrixDense<__half>::operator*(const Vector<__half> &vec) const {

    if (vec.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid vec in matrix-vector prod (operator*(const Vector<__half> &vec))"
        );
    }

    Vector<__half> c(Vector<__half>::Zero(cu_handles, m_rows));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, n_cols,
            SCALAR_ONE_H.d_scalar,
            d_mat, HIP_R_16F, m_rows,
            vec.d_vec, HIP_R_16F, n_cols,
            SCALAR_ZERO_H.d_scalar,
            c.d_vec, HIP_R_16F, m_rows,
            HIPBLAS_COMPUTE_16F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

Vector<__half> MatrixDense<__half>::mult_subset_cols(int start, int cols, const Vector<__half> &vec) const {

    if ((start < 0) || ((start+cols) > n_cols) || (cols <= 0)) {
        throw std::runtime_error("MatrixDense: invalid column values in mult_subset_cols");
    }
    if (vec.rows() != cols) {
        throw std::runtime_error("MatrixDense: invalid vec in mult_subset_cols");
    }

    Vector<__half> c(Vector<__half>::Zero(cu_handles, m_rows));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, cols,
            SCALAR_ONE_H.d_scalar,
            d_mat+start*m_rows, HIP_R_16F, m_rows,
            vec.d_vec, HIP_R_16F, n_cols,
            SCALAR_ZERO_H.d_scalar,
            c.d_vec, HIP_R_16F, m_rows,
            HIPBLAS_COMPUTE_16F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

Vector<__half> MatrixDense<__half>::transpose_prod(const Vector<__half> &vec) const {

    if (vec.rows() != m_rows) { throw std::runtime_error("MatrixDense: invalid vec in transpose_prod"); }

    Vector<__half> c(cu_handles, n_cols);

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_T, HIPBLAS_OP_N,
            n_cols, 1, m_rows,
            SCALAR_ONE_H.d_scalar,
            d_mat, HIP_R_16F, m_rows,
            vec.d_vec, HIP_R_16F, m_rows,
            SCALAR_ZERO_H.d_scalar,
            c.d_vec, HIP_R_16F, n_cols,
            HIPBLAS_COMPUTE_16F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

Vector<__half> MatrixDense<__half>::transpose_prod_subset_cols(int start, int cols, const Vector<__half> &vec) const {

    if ((start < 0) || ((start+cols) > n_cols) || (cols <= 0)) {
        throw std::runtime_error("MatrixDense: invalid column values in transpose_prod_subset_cols");
    }
    if (vec.rows() != m_rows) {
        throw std::runtime_error("MatrixDense: invalid vec in transpose_prod_subset_cols");
    }

    Vector<__half> c(Vector<__half>::Zero(cu_handles, cols));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_T, HIPBLAS_OP_N,
            cols, 1, m_rows,
            SCALAR_ONE_H.d_scalar,
            d_mat+start*m_rows, HIP_R_16F, m_rows,
            vec.d_vec, HIP_R_16F, m_rows,
            SCALAR_ZERO_H.d_scalar,
            c.d_vec, HIP_R_16F, cols,
            HIPBLAS_COMPUTE_16F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

MatrixDense<__half> MatrixDense<__half>::operator*(const MatrixDense<__half> &mat) const {

    if (mat.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix-matrix prod (operator*(const MatrixDense<__half> &mat))"
        );
    }

    MatrixDense<__half> c(MatrixDense<__half>::Zero(cu_handles, m_rows, mat.cols()));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, mat.cols(), n_cols,
            SCALAR_ONE_H.d_scalar,
            d_mat, HIP_R_16F, m_rows,
            mat.d_mat, HIP_R_16F, n_cols,
            SCALAR_ZERO_H.d_scalar,
            c.d_mat, HIP_R_16F, m_rows,
            HIPBLAS_COMPUTE_16F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

MatrixDense<__half> MatrixDense<__half>::operator+(const MatrixDense<__half> &mat) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix add (operator+(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<__half> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            SCALAR_ONE_F.d_scalar, HIP_R_32F,
            mat.d_mat, HIP_R_16F, 1,
            c.d_mat, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return c;

}

MatrixDense<__half> MatrixDense<__half>::operator-(const MatrixDense<__half> &mat) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix subtract (operator-(const MatrixDense<double> &mat))"
        );
    }

    MatrixDense<__half> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            SCALAR_MINUS_ONE_F.d_scalar, HIP_R_32F,
            mat.d_mat, HIP_R_16F, 1,
            c.d_mat, HIP_R_16F, 1,
            HIP_R_32F
        )
    );

    return c;

}

Scalar<__half> MatrixDense<__half>::norm() const {

    Scalar<__half> result;

    check_cublas_status(
        hipblasNrm2Ex(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            d_mat, HIP_R_16F, 1,
            result.d_scalar, HIP_R_16F,
            HIP_R_32F
        )
    );

    return result;

}

MatrixDense<__half> MatrixDense<__half>::to_half() const {
    return MatrixDense<__half>(*this);
}

MatrixDense<float> MatrixDense<__half>::to_float() const {
    
    MatrixDense<float> created_mat(cu_handles, m_rows, n_cols);

    int NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    int NUM_BLOCKS = std::ceil(static_cast<double>(m_rows*n_cols)/static_cast<double>(NUM_THREADS));

    if (NUM_BLOCKS > 0) {

        generalmatrix_hlf_kernels::cast_to_float<<<NUM_BLOCKS, NUM_THREADS>>>(
            d_mat, created_mat.d_mat, m_rows*n_cols
        );

        check_kernel_launch(
            hipGetLastError(),
            "MatrixDense<__half>::to_float",
            "generalmatrix_hlf_kernels::cast_to_float",
            NUM_BLOCKS, NUM_THREADS
        );

    }

    return created_mat;

}

MatrixDense<double> MatrixDense<__half>::to_double() const {
    
    MatrixDense<double> created_mat(cu_handles, m_rows, n_cols);

    int NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    int NUM_BLOCKS = std::ceil(static_cast<double>(m_rows*n_cols)/static_cast<double>(NUM_THREADS));

    if (NUM_BLOCKS > 0) {

        generalmatrix_hlf_kernels::cast_to_double<<<NUM_BLOCKS, NUM_THREADS>>>(
            d_mat, created_mat.d_mat, m_rows*n_cols
        );

        check_kernel_launch(
            hipGetLastError(),
            "MatrixDense<__half>::to_double",
            "generalmatrix_hlf_kernels::cast_to_double",
            NUM_BLOCKS, NUM_THREADS
        );

    }

    return created_mat;

}