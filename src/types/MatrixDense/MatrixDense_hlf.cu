#include "hip/hip_runtime.h"
#include "types/MatrixDense.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixDense<__half> MatrixDense<__half>::operator*(const __half &scalar) const {

    MatrixDense<__half> c(*this);

    float scalar_cast = static_cast<float>(scalar);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows*n_cols, &scalar_cast, HIP_R_32F, c.d_mat, HIP_R_16F, 1, HIP_R_32F
        )
    );

    return c;
}

template <>
MatrixVector<__half> MatrixDense<__half>::operator*(const MatrixVector<__half> &vec) const {

    if (vec.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid vec in matrix-vector prod (operator*(const MatrixVector<__half> &vec))"
        );
    }

    MatrixVector<__half> c(MatrixVector<__half>::Zero(handle, m_rows));

    __half alpha = 1.;
    __half beta = 0.;

    check_cublas_status(
        hipblasGemmEx(
            handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, n_cols,
            &alpha,
            d_mat, HIP_R_16F, m_rows,
            vec.d_vec, HIP_R_16F, n_cols,
            &beta,
            c.d_vec, HIP_R_16F, m_rows,
            HIPBLAS_COMPUTE_16F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

template <>
MatrixVector<__half> MatrixDense<__half>::transpose_prod(const MatrixVector<__half> &vec) const {

    if (vec.rows() != m_rows) { throw std::runtime_error("MatrixDense: invalid vec in transpose_prod"); }

    MatrixVector<__half> c(handle, n_cols);

    __half alpha = static_cast<__half>(1.);
    __half beta = static_cast<__half>(0.);

    check_cublas_status(
        hipblasGemmEx(
            handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            n_cols, 1, m_rows,
            &alpha,
            d_mat, HIP_R_16F, m_rows,
            vec.d_vec, HIP_R_16F, m_rows,
            &beta,
            c.d_vec, HIP_R_16F, n_cols,
            HIPBLAS_COMPUTE_16F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}


template <>
MatrixDense<__half> MatrixDense<__half>::operator*(const MatrixDense<__half> &mat) const {

    if (mat.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix-matrix prod (operator*(const MatrixDense<__half> &mat))"
        );
    }

    MatrixDense<__half> c(MatrixDense<__half>::Zero(handle, m_rows, mat.cols()));

    __half alpha = 1.;
    __half beta = 0.;

    check_cublas_status(
        hipblasGemmEx(
            handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, mat.cols(), n_cols,
            &alpha,
            d_mat, HIP_R_16F, m_rows,
            mat.d_mat, HIP_R_16F, n_cols,
            &beta,
            c.d_mat, HIP_R_16F, m_rows,
            HIPBLAS_COMPUTE_16F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}