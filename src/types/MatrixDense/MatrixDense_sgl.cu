#include "types/MatrixDense.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixDense<float> MatrixDense<float>::operator*(const float &scalar) const {

    MatrixDense<float> c(*this);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows*n_cols, &scalar, HIP_R_32F, c.d_mat, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return c;

}

template <>
MatrixVector<float> MatrixDense<float>::operator*(const MatrixVector<float> &vec) const {

    if (vec.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid vec in matrix-vector prod (operator*(const MatrixVector<float> &vec))"
        );
    }

    MatrixVector<float> c(MatrixVector<float>::Zero(handle, m_rows));

    float alpha = 1.;
    float beta = 0.;

    check_cublas_status(
        hipblasGemmEx(
            handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, n_cols,
            &alpha,
            d_mat, HIP_R_32F, m_rows,
            vec.d_vec, HIP_R_32F, n_cols,
            &beta,
            c.d_vec, HIP_R_32F, m_rows,
            HIPBLAS_COMPUTE_32F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

template <>
MatrixVector<float> MatrixDense<float>::transpose_prod(const MatrixVector<float> &vec) const {

    if (vec.rows() != m_rows) { throw std::runtime_error("MatrixDense: invalid vec in transpose_prod"); }

    MatrixVector<float> c(handle, n_cols);

    float alpha = static_cast<float>(1.);
    float beta = static_cast<float>(0.);

    check_cublas_status(
        hipblasGemmEx(
            handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
            n_cols, 1, m_rows,
            &alpha,
            d_mat, HIP_R_32F, m_rows,
            vec.d_vec, HIP_R_32F, m_rows,
            &beta,
            c.d_vec, HIP_R_32F, n_cols,
            HIPBLAS_COMPUTE_32F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}