#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "types/MatrixDense/MatrixDense.h"

MatrixDense<float> MatrixDense<float>::operator*(const Scalar<float> &scalar) const {

    MatrixDense<float> c(*this);

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            scalar.d_scalar, HIP_R_32F,
            c.d_mat, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return c;

}

MatrixDense<float> & MatrixDense<float>::operator*=(const Scalar<float> &scalar) {

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            scalar.d_scalar, HIP_R_32F,
            d_mat, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

Vector<float> MatrixDense<float>::operator*(const Vector<float> &vec) const {

    if (vec.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid vec in matrix-vector prod (operator*(const Vector<float> &vec))"
        );
    }

    Vector<float> c(Vector<float>::Zero(cu_handles, m_rows));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, n_cols,
            SCALAR_ONE_F.d_scalar,
            d_mat, HIP_R_32F, m_rows,
            vec.d_vec, HIP_R_32F, n_cols,
            SCALAR_ZERO_F.d_scalar,
            c.d_vec, HIP_R_32F, m_rows,
            HIPBLAS_COMPUTE_32F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

Vector<float> MatrixDense<float>::transpose_prod(const Vector<float> &vec) const {

    if (vec.rows() != m_rows) { throw std::runtime_error("MatrixDense: invalid vec in transpose_prod"); }

    Vector<float> c(cu_handles, n_cols);

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_T, HIPBLAS_OP_N,
            n_cols, 1, m_rows,
            SCALAR_ONE_F.d_scalar,
            d_mat, HIP_R_32F, m_rows,
            vec.d_vec, HIP_R_32F, m_rows,
            SCALAR_ZERO_F.d_scalar,
            c.d_vec, HIP_R_32F, n_cols,
            HIPBLAS_COMPUTE_32F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

MatrixDense<float> MatrixDense<float>::operator*(const MatrixDense<float> &mat) const {

    if (mat.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix-matrix prod (operator*(const MatrixDense<float> &mat))"
        );
    }

    MatrixDense<float> c(MatrixDense<float>::Zero(cu_handles, m_rows, mat.cols()));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, mat.cols(), n_cols,
            SCALAR_ONE_F.d_scalar,
            d_mat, HIP_R_32F, m_rows,
            mat.d_mat, HIP_R_32F, n_cols,
            SCALAR_ZERO_F.d_scalar,
            c.d_mat, HIP_R_32F, m_rows,
            HIPBLAS_COMPUTE_32F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

MatrixDense<float> MatrixDense<float>::operator+(const MatrixDense<float> &mat) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix add (operator+(const MatrixDense<float> &mat))"
        );
    }

    MatrixDense<float> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            SCALAR_ONE_F.d_scalar, HIP_R_32F,
            mat.d_mat, HIP_R_32F, 1,
            c.d_mat, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return c;

}

MatrixDense<float> MatrixDense<float>::operator-(const MatrixDense<float> &mat) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix subtract (operator-(const MatrixDense<float> &mat))"
        );
    }

    MatrixDense<float> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            SCALAR_MINUS_ONE_F.d_scalar, HIP_R_32F,
            mat.d_mat, HIP_R_32F, 1,
            c.d_mat, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return c;

}

Scalar<float> MatrixDense<float>::norm() const {

    Scalar<float> result;

    check_cublas_status(
        hipblasNrm2Ex(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            d_mat, HIP_R_32F, 1,
            result.d_scalar, HIP_R_32F,
            HIP_R_32F
        )
    );

    return result;

}

Vector<float> MatrixDense<float>::back_sub(const Vector<float> &arg_rhs) const {

    if (m_rows != n_cols) {
        throw std::runtime_error("MatrixDense::back_sub: non-square matrix");
    }
    if (m_rows != arg_rhs.rows()) {
        throw std::runtime_error("MatrixDense::back_sub: incompatible matrix and rhs");
    }

    Vector<float> soln(arg_rhs);

    float *d_scale_val;
    check_cuda_error(hipMalloc(&d_scale_val, sizeof(float)));

    for (int col=n_cols-1; col>=0; --col) {

        matrixdense_sgl_kernels::solve_pivot_and_find_alpha<<<1, 1>>>(
            soln.d_vec+col, d_mat+(col*m_rows+col), d_scale_val
        );
        if (col > 0) {
            check_cublas_status(
                hipblasAxpyEx(
                    cu_handles.get_cublas_handle(),
                    col,
                    d_scale_val, HIP_R_32F,
                    d_mat+(col*m_rows), HIP_R_32F, 1,
                    soln.d_vec, HIP_R_32F, 1,
                    HIP_R_32F
                )
            );
        }

    }

    check_cuda_error(hipFree(d_scale_val));

    return soln;

}

MatrixDense<__half> MatrixDense<float>::to_half() const {
    
    MatrixDense<__half> created_mat(cu_handles, m_rows, n_cols);

    int NUM_THREADS = 1024; // threads per thread block just maximum
    int NUM_BLOCKS = std::ceil(static_cast<double>(m_rows*n_cols)/static_cast<double>(NUM_THREADS));

    generalmatrix_sgl_kernels::cast_to_half<<<NUM_THREADS, NUM_BLOCKS>>>(
        d_mat, created_mat.d_mat, m_rows*n_cols
    );

    return created_mat;

}

MatrixDense<float> MatrixDense<float>::to_float() const {
    return MatrixDense<float>(*this);
}

MatrixDense<double> MatrixDense<float>::to_double() const {
    
    MatrixDense<double> created_mat(cu_handles, m_rows, n_cols);

    int NUM_THREADS = 1024; // threads per thread block just maximum
    int NUM_BLOCKS = std::ceil(static_cast<double>(m_rows*n_cols)/static_cast<double>(NUM_THREADS));

    generalmatrix_sgl_kernels::cast_to_double<<<NUM_THREADS, NUM_BLOCKS>>>(
        d_mat, created_mat.d_mat, m_rows*n_cols
    );

    return created_mat;

}