#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#include "types/GeneralMatrix/GeneralMatrix_gpu_constants.cuh"

#include "types/MatrixDense/MatrixDense.h"

MatrixDense<float> MatrixDense<float>::operator*(const Scalar<float> &scalar) const {

    MatrixDense<float> c(*this);

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            scalar.d_scalar, HIP_R_32F,
            c.d_mat, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return c;

}

MatrixDense<float> & MatrixDense<float>::operator*=(const Scalar<float> &scalar) {

    check_cublas_status(
        hipblasScalEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            scalar.d_scalar, HIP_R_32F,
            d_mat, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return *this;

}

Vector<float> MatrixDense<float>::operator*(const Vector<float> &vec) const {

    if (vec.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid vec in matrix-vector prod (operator*(const Vector<float> &vec))"
        );
    }

    Vector<float> c(Vector<float>::Zero(cu_handles, m_rows));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, n_cols,
            SCALAR_ONE_F.d_scalar,
            d_mat, HIP_R_32F, m_rows,
            vec.d_vec, HIP_R_32F, n_cols,
            SCALAR_ZERO_F.d_scalar,
            c.d_vec, HIP_R_32F, m_rows,
            HIPBLAS_COMPUTE_32F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

Vector<float> MatrixDense<float>::mult_subset_cols(int start, int cols, const Vector<float> &vec) const {

    if ((start < 0) || ((start+cols) > n_cols) || (cols <= 0)) {
        throw std::runtime_error("MatrixDense: invalid column values in mult_subset_cols");
    }
    if (vec.rows() != cols) {
        throw std::runtime_error("MatrixDense: invalid vec in mult_subset_cols");
    }

    Vector<float> c(Vector<float>::Zero(cu_handles, m_rows));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, 1, cols,
            SCALAR_ONE_F.d_scalar,
            d_mat+start*m_rows, HIP_R_32F, m_rows,
            vec.d_vec, HIP_R_32F, n_cols,
            SCALAR_ZERO_F.d_scalar,
            c.d_vec, HIP_R_32F, m_rows,
            HIPBLAS_COMPUTE_32F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

Vector<float> MatrixDense<float>::transpose_prod(const Vector<float> &vec) const {

    if (vec.rows() != m_rows) { throw std::runtime_error("MatrixDense: invalid vec in transpose_prod"); }

    Vector<float> c(cu_handles, n_cols);

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_T, HIPBLAS_OP_N,
            n_cols, 1, m_rows,
            SCALAR_ONE_F.d_scalar,
            d_mat, HIP_R_32F, m_rows,
            vec.d_vec, HIP_R_32F, m_rows,
            SCALAR_ZERO_F.d_scalar,
            c.d_vec, HIP_R_32F, n_cols,
            HIPBLAS_COMPUTE_32F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

Vector<float> MatrixDense<float>::transpose_prod_subset_cols(int start, int cols, const Vector<float> &vec) const {

    if ((start < 0) || ((start+cols) > n_cols) || (cols <= 0)) {
        throw std::runtime_error("MatrixDense: invalid column values in transpose_prod_subset_cols");
    }
    if (vec.rows() != m_rows) {
        throw std::runtime_error("MatrixDense: invalid vec in transpose_prod_subset_cols");
    }

    Vector<float> c(Vector<float>::Zero(cu_handles, cols));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_T, HIPBLAS_OP_N,
            cols, 1, m_rows,
            SCALAR_ONE_F.d_scalar,
            d_mat+start*m_rows, HIP_R_32F, m_rows,
            vec.d_vec, HIP_R_32F, m_rows,
            SCALAR_ZERO_F.d_scalar,
            c.d_vec, HIP_R_32F, cols,
            HIPBLAS_COMPUTE_32F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

MatrixDense<float> MatrixDense<float>::operator*(const MatrixDense<float> &mat) const {

    if (mat.rows() != n_cols) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix-matrix prod (operator*(const MatrixDense<float> &mat))"
        );
    }

    MatrixDense<float> c(MatrixDense<float>::Zero(cu_handles, m_rows, mat.cols()));

    check_cublas_status(
        hipblasGemmEx(
            cu_handles.get_cublas_handle(),
            HIPBLAS_OP_N, HIPBLAS_OP_N,
            m_rows, mat.cols(), n_cols,
            SCALAR_ONE_F.d_scalar,
            d_mat, HIP_R_32F, m_rows,
            mat.d_mat, HIP_R_32F, n_cols,
            SCALAR_ZERO_F.d_scalar,
            c.d_mat, HIP_R_32F, m_rows,
            HIPBLAS_COMPUTE_32F,
            HIPBLAS_GEMM_DEFAULT
        )
    );

    return c;

}

MatrixDense<float> MatrixDense<float>::operator+(const MatrixDense<float> &mat) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix add (operator+(const MatrixDense<float> &mat))"
        );
    }

    MatrixDense<float> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            SCALAR_ONE_F.d_scalar, HIP_R_32F,
            mat.d_mat, HIP_R_32F, 1,
            c.d_mat, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return c;

}

MatrixDense<float> MatrixDense<float>::operator-(const MatrixDense<float> &mat) const {

    if ((mat.rows() != m_rows) || (mat.cols() != n_cols)) {
        throw std::runtime_error(
            "MatrixDense: invalid mat in matrix subtract (operator-(const MatrixDense<float> &mat))"
        );
    }

    MatrixDense<float> c(*this);

    check_cublas_status(
        hipblasAxpyEx(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            SCALAR_MINUS_ONE_F.d_scalar, HIP_R_32F,
            mat.d_mat, HIP_R_32F, 1,
            c.d_mat, HIP_R_32F, 1,
            HIP_R_32F
        )
    );

    return c;

}

Scalar<float> MatrixDense<float>::norm() const {

    Scalar<float> result;

    check_cublas_status(
        hipblasNrm2Ex(
            cu_handles.get_cublas_handle(),
            m_rows*n_cols,
            d_mat, HIP_R_32F, 1,
            result.d_scalar, HIP_R_32F,
            HIP_R_32F
        )
    );

    return result;

}

MatrixDense<__half> MatrixDense<float>::to_half() const {
    
    MatrixDense<__half> created_mat(cu_handles, m_rows, n_cols);

    int NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    int NUM_BLOCKS = std::ceil(static_cast<double>(m_rows*n_cols)/static_cast<double>(NUM_THREADS));

    if (NUM_BLOCKS > 0) {

        generalmatrix_sgl_kernels::cast_to_half<<<NUM_BLOCKS, NUM_THREADS>>>(
            d_mat, created_mat.d_mat, m_rows*n_cols
        );

        check_kernel_launch(
            hipGetLastError(),
            "MatrixDense<float>::to_half",
            "generalmatrix_sgl_kernels::cast_to_half",
            NUM_BLOCKS, NUM_THREADS
        );

    }

    return created_mat;

}

MatrixDense<float> MatrixDense<float>::to_float() const {
    return MatrixDense<float>(*this);
}

MatrixDense<double> MatrixDense<float>::to_double() const {
    
    MatrixDense<double> created_mat(cu_handles, m_rows, n_cols);

    int NUM_THREADS = genmat_gpu_const::MAXTHREADSPERBLOCK;
    int NUM_BLOCKS = std::ceil(static_cast<double>(m_rows*n_cols)/static_cast<double>(NUM_THREADS));

    if (NUM_BLOCKS > 0) {

        generalmatrix_sgl_kernels::cast_to_double<<<NUM_BLOCKS, NUM_THREADS>>>(
            d_mat, created_mat.d_mat, m_rows*n_cols
        );

        check_kernel_launch(
            hipGetLastError(),
            "MatrixDense<float>::to_double",
            "generalmatrix_sgl_kernels::cast_to_double",
            NUM_THREADS, NUM_BLOCKS
        );

    }

    return created_mat;

}