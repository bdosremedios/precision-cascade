#include "types/MatrixDense.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixDense<float> MatrixDense<float>::operator*(const float &scalar) const {

    MatrixDense<float> c(*this);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows*n_cols, &scalar, HIP_R_32F, c.d_mat, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return c;
}