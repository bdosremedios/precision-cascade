#include "hip/hip_runtime.h"
#include "types/MatrixVector.h"
#include "tools/cublas_check.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixVector<__half> MatrixVector<__half>::operator*(const __half &scalar) const {

    MatrixVector<__half> c(*this);
    float *scalar_cast = static_cast<float *>(malloc(sizeof(float)));
    *scalar_cast = static_cast<float>(scalar);

    hipblasStatus_t status = hipblasScalEx(
        handle, m, scalar_cast, HIP_R_32F, c.d_vec, HIP_R_16F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    free(scalar_cast);

    return c;

}

template<>
MatrixVector<__half>& MatrixVector<__half>::operator*=(const __half &scalar) {

    float *scalar_cast = static_cast<float *>(malloc(sizeof(float)));
    *scalar_cast = static_cast<float>(scalar);

    hipblasStatus_t status = hipblasScalEx(
        handle, m, scalar_cast, HIP_R_32F, d_vec, HIP_R_16F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    free(scalar_cast);

    return *this;

}