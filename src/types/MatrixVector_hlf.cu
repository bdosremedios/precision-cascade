#include "hip/hip_runtime.h"
#include "types/MatrixVector.h"
#include "tools/cublas_check.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixVector<__half> MatrixVector<__half>::operator*(const __half &scalar) const {

    MatrixVector<__half> c(*this);
    float *scalar_cast = static_cast<float *>(malloc(sizeof(float)));
    *scalar_cast = static_cast<float>(scalar);

    hipblasStatus_t status = hipblasScalEx(
        handle, m, scalar_cast, HIP_R_32F, c.d_vec, HIP_R_16F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    free(scalar_cast);

    return c;

}

template<>
MatrixVector<__half>& MatrixVector<__half>::operator*=(const __half &scalar) {

    float scalar_cast = static_cast<float>(scalar);

    hipblasStatus_t status = hipblasScalEx(
        handle, m, &scalar_cast, HIP_R_32F, d_vec, HIP_R_16F, 1, HIP_R_32F
    );
    check_cublas_status(status);


    return *this;

}

template<>
MatrixVector<__half> MatrixVector<__half>::operator+(const MatrixVector<__half> &vec) const {

    MatrixVector<__half> c(*this);
    float alpha = 1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, c.d_vec, HIP_R_16F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<__half> MatrixVector<__half>::operator-(const MatrixVector<__half> &vec) const {

    MatrixVector<__half> c(*this);
    float alpha = -1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, c.d_vec, HIP_R_16F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<__half>& MatrixVector<__half>::operator+=(const MatrixVector<__half> &vec) {

    float alpha = 1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, d_vec, HIP_R_16F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return *this;

}

template<>
MatrixVector<__half>& MatrixVector<__half>::operator-=(const MatrixVector<__half> &vec) {

    float alpha = -1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_32F, vec.d_vec, HIP_R_16F, 1, d_vec, HIP_R_16F, 1, HIP_R_32F
    );
    check_cublas_status(status);

    return *this;

}

template<>
__half MatrixVector<__half>::dot(const MatrixVector<__half> &vec) const {
    
    __half result;

    hipblasStatus_t status = hipblasDotEx(
        handle, m, d_vec, HIP_R_16F, 1, vec.d_vec, HIP_R_16F, 1, &result, HIP_R_16F, HIP_R_32F
    );
    check_cublas_status(status);

    return result;

}

template<>
__half MatrixVector<__half>::norm() const {

    __half result;

    hipblasStatus_t status = hipblasNrm2Ex(
        handle, m, d_vec, HIP_R_16F, 1, &result, HIP_R_16F, HIP_R_32F
    );
    check_cublas_status(status);

    return result;

}