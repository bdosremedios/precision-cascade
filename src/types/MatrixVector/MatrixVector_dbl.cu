#include "hip/hip_runtime.h"
#include "types/MatrixVector.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixVector<double> MatrixVector<double>::operator*(const double &scalar) const {

    MatrixVector<double> c(*this);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, &scalar, HIP_R_64F, c.d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return c;

}

template<>
MatrixVector<double> & MatrixVector<double>::operator*=(const double &scalar) {

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, &scalar, HIP_R_64F, d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return *this;

}

template<>
MatrixVector<double> MatrixVector<double>::operator+(const MatrixVector<double> &vec) const {

    check_vecvec_op_compatibility(vec);

    MatrixVector<double> c(*this);
    double alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, c.d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return c;

}

template<>
MatrixVector<double> MatrixVector<double>::operator-(const MatrixVector<double> &vec) const {

    check_vecvec_op_compatibility(vec);

    MatrixVector<double> c(*this);
    double alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, c.d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return c;

}

template<>
MatrixVector<double> & MatrixVector<double>::operator+=(const MatrixVector<double> &vec) {

    check_vecvec_op_compatibility(vec);

    double alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return *this;

}

template<>
MatrixVector<double> & MatrixVector<double>::operator-=(const MatrixVector<double> &vec) {

    check_vecvec_op_compatibility(vec);

    double alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return *this;

}

template<>
double MatrixVector<double>::dot(const MatrixVector<double> &vec) const {

    check_vecvec_op_compatibility(vec);
    
    double result;

    check_cublas_status(
        hipblasDotEx(
            handle, m_rows, d_vec, HIP_R_64F, 1, vec.d_vec, HIP_R_64F, 1, &result, HIP_R_64F, HIP_R_64F
        )
    );

    return result;

}

template<>
double MatrixVector<double>::norm() const {

    double result;

    check_cublas_status(
        hipblasNrm2Ex(
            handle, m_rows, d_vec, HIP_R_64F, 1, &result, HIP_R_64F, HIP_R_64F
        )
    );

    return result;

}