#include "hip/hip_runtime.h"
#include "types/MatrixVector.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixVector<double> MatrixVector<double>::operator*(const double &scalar) const {

    MatrixVector<double> c(*this);

    check_cublas_status(
        hipblasScalEx(
            handle, m, &scalar, HIP_R_64F, c.d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return c;

}

template<>
MatrixVector<double> & MatrixVector<double>::operator*=(const double &scalar) {

    check_cublas_status(
        hipblasScalEx(
            handle, m, &scalar, HIP_R_64F, d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return *this;

}

template<>
MatrixVector<double> MatrixVector<double>::operator+(const MatrixVector<double> &vec) const {

    MatrixVector<double> c(*this);
    double alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, c.d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return c;

}

template<>
MatrixVector<double> MatrixVector<double>::operator-(const MatrixVector<double> &vec) const {

    MatrixVector<double> c(*this);
    double alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, c.d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return c;

}

template<>
MatrixVector<double> & MatrixVector<double>::operator+=(const MatrixVector<double> &vec) {

    double alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return *this;

}

template<>
MatrixVector<double> & MatrixVector<double>::operator-=(const MatrixVector<double> &vec) {

    double alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, d_vec, HIP_R_64F, 1, HIP_R_64F
        )
    );

    return *this;

}

template<>
double MatrixVector<double>::dot(const MatrixVector<double> &vec) const {
    
    double result;

    check_cublas_status(
        hipblasDotEx(
            handle, m, d_vec, HIP_R_64F, 1, vec.d_vec, HIP_R_64F, 1, &result, HIP_R_64F, HIP_R_64F
        )
    );

    return result;

}

template<>
double MatrixVector<double>::norm() const {

    double result;

    check_cublas_status(
        hipblasNrm2Ex(
            handle, m, d_vec, HIP_R_64F, 1, &result, HIP_R_64F, HIP_R_64F
        )
    );

    return result;

}