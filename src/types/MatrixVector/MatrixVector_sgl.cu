#include "hip/hip_runtime.h"
#include "types/MatrixVector.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixVector<float> MatrixVector<float>::operator*(const float &scalar) const {

    MatrixVector<float> c(*this);

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, &scalar, HIP_R_32F, c.d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return c;

}

template<>
MatrixVector<float> & MatrixVector<float>::operator*=(const float &scalar) {

    check_cublas_status(
        hipblasScalEx(
            handle, m_rows, &scalar, HIP_R_32F, d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return *this;

}

template<>
MatrixVector<float> MatrixVector<float>::operator+(const MatrixVector<float> &vec) const {

    check_vecvec_op_compatibility(vec);

    MatrixVector<float> c(*this);
    float alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, c.d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return c;

}

template<>
MatrixVector<float> MatrixVector<float>::operator-(const MatrixVector<float> &vec) const {

    check_vecvec_op_compatibility(vec);

    MatrixVector<float> c(*this);
    float alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, c.d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return c;

}

template<>
MatrixVector<float> & MatrixVector<float>::operator+=(const MatrixVector<float> &vec) {

    check_vecvec_op_compatibility(vec);

    float alpha = 1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return *this;

}

template<>
MatrixVector<float> & MatrixVector<float>::operator-=(const MatrixVector<float> &vec) {

    check_vecvec_op_compatibility(vec);

    float alpha = -1.;

    check_cublas_status(
        hipblasAxpyEx(
            handle, m_rows, &alpha, HIP_R_32F, vec.d_vec, HIP_R_32F, 1, d_vec, HIP_R_32F, 1, HIP_R_32F
        )
    );

    return *this;

}

template<>
float MatrixVector<float>::dot(const MatrixVector<float> &vec) const {

    check_vecvec_op_compatibility(vec);
    
    float result;

    check_cublas_status(
        hipblasDotEx(
            handle, m_rows, d_vec, HIP_R_32F, 1, vec.d_vec, HIP_R_32F, 1, &result, HIP_R_32F, HIP_R_32F
        )
    );

    return result;

}

template<>
float MatrixVector<float>::norm() const {

    float result;

    check_cublas_status(
        hipblasNrm2Ex(
            handle, m_rows, d_vec, HIP_R_32F, 1, &result, HIP_R_32F, HIP_R_32F
        )
    );

    return result;

}