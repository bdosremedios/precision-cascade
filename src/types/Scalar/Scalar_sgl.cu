#include "hip/hip_runtime.h"
#include "types/Scalar/Scalar.h"
#include "types/Scalar/Scalar_template_subroutines.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

template Scalar<float> Scalar<float>::operator+(const Scalar<float> &other) const;
template Scalar<float> Scalar<float>::operator-(const Scalar<float> &other) const;

template Scalar<float> & Scalar<float>::operator+=(const Scalar<float> &);
template Scalar<float> & Scalar<float>::operator-=(const Scalar<float> &);

template Scalar<float> Scalar<float>::operator*(const Scalar<float> &) const;
template Scalar<float> Scalar<float>::operator/(const Scalar<float> &) const;

template Scalar<float> & Scalar<float>::operator*=(const Scalar<float> &);
template Scalar<float> & Scalar<float>::operator/=(const Scalar<float> &);

template bool Scalar<float>::operator==(const Scalar<float> &) const;

Scalar<float> & Scalar<float>::abs() {
    scalar_sgl_kernels::scalar_abs<<<1, 1>>>(d_scalar);
    check_kernel_launch(
        hipGetLastError(),
        "Scalar<float>::abs",
        "scalar_sgl_kernels::scalar_abs",
        1, 1
    );
    return *this;
}

Scalar<float> & Scalar<float>::sqrt() {
    scalar_sgl_kernels::scalar_sqrt<<<1, 1>>>(d_scalar);
    check_kernel_launch(
        hipGetLastError(),
        "Scalar<float>::sqrt",
        "scalar_sgl_kernels::scalar_sqrt",
        1, 1
    );
    return *this;
}

Scalar<float> & Scalar<float>::reciprocol() {
    scalar_sgl_kernels::scalar_recip<<<1, 1>>>(d_scalar);
    check_kernel_launch(
        hipGetLastError(),
        "Scalar<float>::reciprocol",
        "scalar_sgl_kernels::scalar_recip",
        1, 1
    );
    return *this;
}

Scalar<__half> Scalar<float>::to_half() const {
    Scalar<__half> created_scalar;
    scalar_sgl_kernels::cast_to_half<<<1, 1>>>(d_scalar, created_scalar.d_scalar);
    check_kernel_launch(
        hipGetLastError(),
        "Scalar<float>::to_half",
        "scalar_sgl_kernels::cast_to_half",
        1, 1
    );
    return created_scalar;
}

Scalar<float> Scalar<float>::to_float() const { return Scalar<float>(*this); }

Scalar<double> Scalar<float>::to_double() const{
    Scalar<double> created_scalar;
    scalar_sgl_kernels::cast_to_double<<<1, 1>>>(d_scalar, created_scalar.d_scalar);
    check_kernel_launch(
        hipGetLastError(),
        "Scalar<float>::to_double",
        "scalar_sgl_kernels::cast_to_double",
        1, 1
    );
    return created_scalar;
}