#include "hip/hip_runtime.h"
#include "types/Scalar/Scalar_gpu_kernels.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_math_constants.h>

// *** Scalar double kernel implementations *** 

__global__ void scalar_dbl_kernels::scalar_abs(double *scalar) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar[tid] = fabs(scalar[tid]);
}

__global__ void scalar_dbl_kernels::scalar_sqrt(double *scalar) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar[tid] = sqrt(scalar[tid]);
}

__global__ void scalar_dbl_kernels::scalar_recip(double *scalar) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar[tid] = HIP_ONE/(scalar[tid]);
}

__global__ void scalar_dbl_kernels::cast_to_half(double *scalar_src, __half *scalar_dest) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar_dest[tid] = __double2half(scalar_src[tid]);
}

__global__ void scalar_dbl_kernels::cast_to_float(double *scalar_src, float *scalar_dest) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar_dest[tid] = __double2float_rn(scalar_src[tid]);
}

// *** Scalar single kernel implementations *** 

__global__ void scalar_sgl_kernels::scalar_abs(float *scalar) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar[tid] = fabsf(scalar[tid]);
}

__global__ void scalar_sgl_kernels::scalar_sqrt(float *scalar) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar[tid] = sqrtf(scalar[tid]);
}

__global__ void scalar_sgl_kernels::scalar_recip(float *scalar) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar[tid] = HIP_ONE_F/(scalar[tid]);
}

__global__ void scalar_sgl_kernels::cast_to_half(float *scalar_src, __half *scalar_dest) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar_dest[tid] = __float2half(scalar_src[tid]);
}

__global__ void scalar_sgl_kernels::cast_to_double(float *scalar_src, double *scalar_dest) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar_dest[tid] = static_cast<double>(scalar_src[tid]);
}

// *** Scalar half kernel implementations ***

__global__ void scalar_hlf_kernels::scalar_abs(half *scalar) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar[tid] = __habs(scalar[tid]);
}

__global__ void scalar_hlf_kernels::scalar_sqrt(half *scalar) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar[tid] = hsqrt(scalar[tid]);
}

__global__ void scalar_hlf_kernels::scalar_recip(__half *scalar) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar[tid] = HIPRT_ONE_FP16/(scalar[tid]);
}

__global__ void scalar_hlf_kernels::cast_to_float(__half *scalar_src, float *scalar_dest) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar_dest[tid] = __half2float(scalar_src[tid]);
}

__global__ void scalar_hlf_kernels::cast_to_double(__half *scalar_src, double *scalar_dest) {
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    scalar_dest[tid] = static_cast<double>(scalar_src[tid]);
}