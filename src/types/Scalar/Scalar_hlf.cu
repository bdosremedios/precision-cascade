#include "hip/hip_runtime.h"
#include "types/Scalar.h"
#include "Scalar.cuh"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template Scalar<__half> Scalar<__half>::operator*(const Scalar<__half> &) const;
template Scalar<__half> Scalar<__half>::operator/(const Scalar<__half> &) const;

template void Scalar<__half>::operator*=(const Scalar<__half> &);
template void Scalar<__half>::operator/=(const Scalar<__half> &);