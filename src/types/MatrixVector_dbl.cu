#include "types/MatrixVector.h"
#include "tools/cublas_check.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixVector<double> MatrixVector<double>::operator*(const double &scalar) const {

    MatrixVector<double> c(*this);

    hipblasStatus_t status = hipblasScalEx(
        handle, m, &scalar, HIP_R_64F, c.d_vec, HIP_R_64F, 1, HIP_R_64F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<double>& MatrixVector<double>::operator*=(const double &scalar) {

    hipblasStatus_t status = hipblasScalEx(
        handle, m, &scalar, HIP_R_64F, d_vec, HIP_R_64F, 1, HIP_R_64F
    );
    check_cublas_status(status);

    return *this;

}