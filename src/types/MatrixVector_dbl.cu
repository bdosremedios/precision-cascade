#include "hip/hip_runtime.h"
#include "types/MatrixVector.h"
#include "tools/cublas_check.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

template<>
MatrixVector<double> MatrixVector<double>::operator*(const double &scalar) const {

    MatrixVector<double> c(*this);

    hipblasStatus_t status = hipblasScalEx(
        handle, m, &scalar, HIP_R_64F, c.d_vec, HIP_R_64F, 1, HIP_R_64F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<double>& MatrixVector<double>::operator*=(const double &scalar) {

    hipblasStatus_t status = hipblasScalEx(
        handle, m, &scalar, HIP_R_64F, d_vec, HIP_R_64F, 1, HIP_R_64F
    );
    check_cublas_status(status);

    return *this;

}

template<>
MatrixVector<double> MatrixVector<double>::operator+(const MatrixVector<double> &vec) const {

    MatrixVector<double> c(*this);
    double alpha = 1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, c.d_vec, HIP_R_64F, 1, HIP_R_64F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<double> MatrixVector<double>::operator-(const MatrixVector<double> &vec) const {

    MatrixVector<double> c(*this);
    double alpha = -1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, c.d_vec, HIP_R_64F, 1, HIP_R_64F
    );
    check_cublas_status(status);

    return c;

}

template<>
MatrixVector<double>& MatrixVector<double>::operator+=(const MatrixVector<double> &vec) {

    double alpha = 1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, d_vec, HIP_R_64F, 1, HIP_R_64F
    );
    check_cublas_status(status);

    return *this;

}

template<>
MatrixVector<double>& MatrixVector<double>::operator-=(const MatrixVector<double> &vec) {

    double alpha = -1.;

    hipblasStatus_t status = hipblasAxpyEx(
        handle, m, &alpha, HIP_R_64F, vec.d_vec, HIP_R_64F, 1, d_vec, HIP_R_64F, 1, HIP_R_64F
    );
    check_cublas_status(status);

    return *this;

}

template<>
double MatrixVector<double>::dot(const MatrixVector<double> &vec) const {
    
    double result;

    hipblasStatus_t status = hipblasDotEx(
        handle, m, d_vec, HIP_R_64F, 1, vec.d_vec, HIP_R_64F, 1, &result, HIP_R_64F, HIP_R_64F
    );
    check_cublas_status(status);

    return result;

}

template<>
double MatrixVector<double>::norm() const {

    double result;

    hipblasStatus_t status = hipblasNrm2Ex(
        handle, m, d_vec, HIP_R_64F, 1, &result, HIP_R_64F, HIP_R_64F
    );
    check_cublas_status(status);

    return result;

}