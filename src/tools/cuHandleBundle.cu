#include "tools/cuHandleBundle.h"

hipblasHandle_t cuHandleBundle::cublas_handle;

hipsparseHandle_t cuHandleBundle::cusparse_handle;

void cuHandleBundle::create() {
    check_cublas_status(hipblasCreate(&cublas_handle));
    check_cublas_status(
        hipblasSetPointerMode(
            cublas_handle,
            HIPBLAS_POINTER_MODE_DEVICE
        )
    );
    check_cusparse_status(hipsparseCreate(&cusparse_handle));
    check_cusparse_status(
        hipsparseSetPointerMode(
            cusparse_handle,
            HIPSPARSE_POINTER_MODE_DEVICE
        )
    );
}

void cuHandleBundle::destroy() {
    check_cublas_status(hipblasDestroy(cublas_handle));
    check_cusparse_status(hipsparseDestroy(cusparse_handle));
}

hipblasHandle_t cuHandleBundle::get_cublas_handle() const {
    return cublas_handle;
}

hipsparseHandle_t cuHandleBundle::get_cusparse_handle() const {
    return cusparse_handle;
}

bool cuHandleBundle::operator==(const cuHandleBundle &other) const {
    return (
        (cublas_handle == other.cublas_handle) &&
        (cusparse_handle == other.cusparse_handle)
    );
}