#include "tools/cuHandleBundle.h"

hipblasHandle_t cascade::cuHandleBundle::cublas_handle;

hipsparseHandle_t cascade::cuHandleBundle::cusparse_handle;

void cascade::cuHandleBundle::create() {
    check_cublas_status(hipblasCreate(&cublas_handle));
    check_cublas_status(
        hipblasSetPointerMode(
            cublas_handle,
            HIPBLAS_POINTER_MODE_DEVICE
        )
    );
    check_cusparse_status(hipsparseCreate(&cusparse_handle));
    check_cusparse_status(
        hipsparseSetPointerMode(
            cusparse_handle,
            HIPSPARSE_POINTER_MODE_DEVICE
        )
    );
}

void cascade::cuHandleBundle::destroy() {
    check_cublas_status(hipblasDestroy(cublas_handle));
    check_cusparse_status(hipsparseDestroy(cusparse_handle));
}

hipblasHandle_t cascade::cuHandleBundle::get_cublas_handle() const {
    return cublas_handle;
}

hipsparseHandle_t cascade::cuHandleBundle::get_cusparse_handle() const {
    return cusparse_handle;
}

bool cascade::cuHandleBundle::operator==(const cuHandleBundle &other) const {
    return (
        (cublas_handle == other.cublas_handle) &&
        (cusparse_handle == other.cusparse_handle)
    );
}