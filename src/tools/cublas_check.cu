#include "tools/cublas_check.h"

#include <stdexcept>
#include <string>

void check_cublas_status(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("cublas error: " + std::to_string(status));
    }
}