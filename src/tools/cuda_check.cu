#include <stdexcept>
#include <string>
#include <iostream>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "tools/cuda_check.h"

void check_cublas_status(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("hipblasStatus_t failure: " + std::to_string(status));
    }
}

void check_cuda_error(hipError_t error) {
    if (error != hipSuccess) {
        throw std::runtime_error("hipError_t failure: " + std::to_string(error));
    }
}