#include "tools/cuda_check.h"

#include <stdexcept>
#include <string>
#include <iostream>

void check_cublas_status(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("hipblasStatus_t failure: " + std::to_string(status));
    }
}

void check_cuda_error(hipError_t error) {
    if (error != hipSuccess) {
        throw std::runtime_error("hipError_t failure: " + std::to_string(error));
    }
}