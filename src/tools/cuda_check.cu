#include <stdexcept>
#include <string>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "tools/cuda_check.h"

void check_cuda_error(hipError_t error) {
    if (error != hipSuccess) {
        throw std::runtime_error("hipError_t failure: " + std::to_string(error));
    }
}

void check_cublas_status(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error("hipblasStatus_t failure: " + std::to_string(status));
    }
}

void check_cusparse_status(hipsparseStatus_t status) {
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        throw std::runtime_error("hipsparseStatus_t failure: " + std::to_string(status));
    }
}