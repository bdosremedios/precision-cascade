#include "tools/cuda_check.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdexcept>
#include <string>

void check_cuda_error(hipError_t error) {
    if (error != hipSuccess) {
        throw std::runtime_error(
            "hipError_t " + std::to_string(static_cast<int>(error)) +
            ": " + hipGetErrorName(error) + " " + hipGetErrorString(error)
        );
    }
}

void check_kernel_launch(
    hipError_t error,
    std::string function_name,
    std::string kernel_name,
    int n_blocks,
    int n_threads
) {
    if (error != hipSuccess) {
        throw std::runtime_error(
            "cuda kernel " + kernel_name +
            "<<<" + std::to_string(n_blocks) + ", " +
            std::to_string(n_threads) + ">>> in " +
            function_name + " failed with error " +
            std::to_string(static_cast<int>(error)) +
            "(" + hipGetErrorName(error) + ": " +
            hipGetErrorString(error) + ")"
        );
    }
}

void check_cublas_status(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error(
            "hipblasStatus_t failure: " +
            std::to_string(static_cast<int>(status))
        );
    }
}

void check_cusparse_status(hipsparseStatus_t status) {
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        throw std::runtime_error(
            "hipsparseStatus_t failure: " +
            std::to_string(static_cast<int>(status))
        );
    }
}