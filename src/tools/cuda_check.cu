#include "tools/cuda_check.h"

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include <stdexcept>
#include <format>
#include <string>

void check_cuda_error(hipError_t error) {
    if (error != hipSuccess) {
        throw std::runtime_error(
            std::format(
                "hipError_t {:d}: {} {}",
                static_cast<int>(error),
                hipGetErrorName(error),
                hipGetErrorString(error)
            )
        );
    }
}

void check_kernel_launch(
    hipError_t error,
    std::string function_name,
    std::string kernel_name,
    int n_blocks,
    int n_threads
) {
    if (error != hipSuccess) {
        throw std::runtime_error(
            std::format(
                "cuda kernel {}<<<{}, {}>>> in {} failed with error {} ({}: {})",
                kernel_name,
                n_blocks, n_threads,
                function_name,
                static_cast<int>(error),
                hipGetErrorName(error),
                hipGetErrorString(error)
            )
        );
    }
}

void check_cublas_status(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error(std::format("hipblasStatus_t failure: {:d}", static_cast<int>(status)));
    }
}

void check_cusparse_status(hipsparseStatus_t status) {
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        throw std::runtime_error(std::format("hipsparseStatus_t failure: {:d}", static_cast<int>(status)));
    }
}