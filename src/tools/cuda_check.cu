#include <stdexcept>
#include <format>

#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "tools/cuda_check.h"

void check_cuda_error(hipError_t error) {
    if (error != hipSuccess) {
        throw std::runtime_error(std::format("hipError_t failure: {:d}", static_cast<int>(error)));
    }
}

void check_cublas_status(hipblasStatus_t status) {
    if (status != HIPBLAS_STATUS_SUCCESS) {
        throw std::runtime_error(std::format("hipblasStatus_t failure: {:d}", static_cast<int>(status)));
    }
}

void check_cusparse_status(hipsparseStatus_t status) {
    if (status != HIPSPARSE_STATUS_SUCCESS) {
        throw std::runtime_error(std::format("hipsparseStatus_t failure: {:d}", static_cast<int>(status)));
    }
}