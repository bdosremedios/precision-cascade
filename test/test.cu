#include "hip/hip_runtime.h"
#include <gtest/gtest.h>
#include <Eigen/Dense>

#include "test.h"

#include <cmath>
#include <string>
#include <iostream>

using Eigen::Matrix, Eigen::Dynamic;

using std::abs;
using std::string;
using std::cout, std::endl;

bool *TestBase::show_plots = new bool;
hipblasHandle_t *TestBase::handle_ptr = new hipblasHandle_t;

int main(int argc, char **argv) {

    testing::InitGoogleTest();

    // Check if should run long tests
    bool run_long_tests = false;
    for (int i=0; i<argc; ++i) {
        if ((string(argv[i]) == "--run_long_tests") || (string(argv[i]) == "-rlt")) { run_long_tests = true; }
    }
    if (run_long_tests) {
        cout << "Running long tests..." << endl;
    } else {
        cout << "Skipping long tests..." << endl;
        testing::GTEST_FLAG(filter) = "-*LONGRUNTIME";
    }

    // Check if should run long tests
    bool only_new = false;
    for (int i=0; i<argc; ++i) {
        if ((string(argv[i]) == "--only-new") || (string(argv[i]) == "-on")) { only_new = true; }
    }
    if (only_new) {
        cout << "Running only new tests..." << endl;
        testing::GTEST_FLAG(filter) = "*NEW";
    } else {
        cout << "Running all tests..." << endl;
    }

    // Check if should show plots
    bool show_plots = false;
    for (int i=0; i<argc; ++i) {
        if ((string(argv[i]) == "--show_plots") || (string(argv[i]) == "-sp")) { show_plots = true; }
    }
    if (show_plots) {
        cout << "Showing plots..." << endl;
        *(TestBase::show_plots) = true;
    } else {
        cout << "Not showing plots..." << endl;
        *(TestBase::show_plots) = false;
    }

    hipblasCreate(TestBase::handle_ptr);
    int return_status = RUN_ALL_TESTS();
    hipblasDestroy(*TestBase::handle_ptr);

    return return_status;

}